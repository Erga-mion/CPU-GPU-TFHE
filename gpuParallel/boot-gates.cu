#include "hip/hip_runtime.h"
#ifndef TFHE_TEST_ENVIRONMENT

#include <cstdlib>
#include <iostream>
#include <random>
#include <cassert>
#include "tfhe_core.h"
#include "numeric_functions.h"
#include "lweparams.h"
#include "lwekey.h"
#include "lwesamples.h"
#include "lwekeyswitch.h"
#include "lwe-functions.h"
#include "lwebootstrappingkey.h"
#include "tfhe.h"
#include <fstream>


using namespace std;

#else
#undef EXPORT
#define EXPORT static
#endif


#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

//*//*****************************************
// zones on the torus -> to see
//*//*****************************************


/*
 * Homomorphic bootstrapped NAND gate
 * Takes in input 2 LWE samples (with message space [-1/8,1/8], noise<1/16)
 * Outputs a LWE bootstrapped sample (with message space [-1/8,1/8], noise<1/16)
*/
EXPORT void
bootsNAND(LweSample *result, const LweSample *ca, const LweSample *cb, const TFheGateBootstrappingCloudKeySet *bk) {
    static const Torus32 MU = modSwitchToTorus32(1, 8);
    const LweParams *in_out_params = bk->params->in_out_params;

    LweSample *temp_result = new_LweSample(in_out_params);

    //compute: (0,1/8) - ca - cb
    static const Torus32 NandConst = modSwitchToTorus32(1, 8);
    lweNoiselessTrivial(temp_result, NandConst, in_out_params);
    lweSubTo(temp_result, ca, in_out_params);
    lweSubTo(temp_result, cb, in_out_params);

    //if the phase is positive, the result is 1/8
    //if the phase is positive, else the result is -1/8
    tfhe_bootstrap_FFT(result, bk->bkFFT, MU, temp_result);

    delete_LweSample(temp_result);
}


/*
 * Homomorphic bootstrapped OR gate
 * Takes in input 2 LWE samples (with message space [-1/8,1/8], noise<1/16)
 * Outputs a LWE bootstrapped sample (with message space [-1/8,1/8], noise<1/16)
*/
EXPORT void
bootsOR(LweSample *result, const LweSample *ca, const LweSample *cb, const TFheGateBootstrappingCloudKeySet *bk) {
    static const Torus32 MU = modSwitchToTorus32(1, 8);
    const LweParams *in_out_params = bk->params->in_out_params;

    LweSample *temp_result = new_LweSample(in_out_params);

    //compute: (0,1/8) + ca + cb
    static const Torus32 OrConst = modSwitchToTorus32(1, 8);
    lweNoiselessTrivial(temp_result, OrConst, in_out_params);
    lweAddTo(temp_result, ca, in_out_params);
    lweAddTo(temp_result, cb, in_out_params);

    //if the phase is positive, the result is 1/8
    //if the phase is positive, else the result is -1/8
    tfhe_bootstrap_FFT(result, bk->bkFFT, MU, temp_result);

    delete_LweSample(temp_result);
}


/*
 * Homomorphic bootstrapped AND gate
 * Takes in input 2 LWE samples (with message space [-1/8,1/8], noise<1/16)
 * Outputs a LWE bootstrapped sample (with message space [-1/8,1/8], noise<1/16)
*/
EXPORT void
bootsAND(LweSample *result, const LweSample *ca, const LweSample *cb, const TFheGateBootstrappingCloudKeySet *bk) {
    static const Torus32 MU = modSwitchToTorus32(1, 8);
    const LweParams *in_out_params = bk->params->in_out_params;

    LweSample *temp_result = new_LweSample(in_out_params);

    //compute: (0,-1/8) + ca + cb
    static const Torus32 AndConst = modSwitchToTorus32(-1, 8);
    lweNoiselessTrivial(temp_result, AndConst, in_out_params);

    lweAddTo(temp_result, ca, in_out_params);
    lweAddTo(temp_result, cb, in_out_params);

//    cout << "old: ";
//    for (int i = 0; i < 10; ++i) {
//        cout << temp_result->a[i] << " ";
//    }
//    cout << endl;

    //if the phase is positive, the result is 1/8
    //if the phase is positive, else the result is -1/8
    tfhe_bootstrap_FFT(result, bk->bkFFT, MU, temp_result);

//    cout << "old: ";
//    for (int i = 0; i < 10; ++i) {
//        cout << result->a[i] << " ";
//    }
//    cout << result->b;
//    cout << endl;

    delete_LweSample(temp_result);
}


/*
 * Homomorphic bootstrapped XOR gate
 * Takes in input 2 LWE samples (with message space [-1/8,1/8], noise<1/16)
 * Outputs a LWE bootstrapped sample (with message space [-1/8,1/8], noise<1/16)
*/
EXPORT void
bootsXOR(LweSample *result, const LweSample *ca, const LweSample *cb, const TFheGateBootstrappingCloudKeySet *bk) {
    static const Torus32 MU = modSwitchToTorus32(1, 8);
    const LweParams *in_out_params = bk->params->in_out_params;

    LweSample *temp_result = new_LweSample(in_out_params);

    //compute: (0,1/4) + 2*(ca + cb)
    static const Torus32 XorConst = modSwitchToTorus32(1, 4);
    lweNoiselessTrivial(temp_result, XorConst, in_out_params);
    lweAddMulTo(temp_result, 2, ca, in_out_params);
    lweAddMulTo(temp_result, 2, cb, in_out_params);

    //if the phase is positive, the result is 1/8
    //if the phase is positive, else the result is -1/8
    tfhe_bootstrap_FFT(result, bk->bkFFT, MU, temp_result);

    delete_LweSample(temp_result);
}


/*
 * Homomorphic bootstrapped XNOR gate
 * Takes in input 2 LWE samples (with message space [-1/8,1/8], noise<1/16)
 * Outputs a LWE bootstrapped sample (with message space [-1/8,1/8], noise<1/16)
*/
EXPORT void
bootsXNOR(LweSample *result, const LweSample *ca, const LweSample *cb, const TFheGateBootstrappingCloudKeySet *bk) {
    static const Torus32 MU = modSwitchToTorus32(1, 8);
    const LweParams *in_out_params = bk->params->in_out_params;

    LweSample *temp_result = new_LweSample(in_out_params);

    //compute: (0,-1/4) + 2*(-ca-cb)
    static const Torus32 XnorConst = modSwitchToTorus32(-1, 4);
    lweNoiselessTrivial(temp_result, XnorConst, in_out_params);
    lweSubMulTo(temp_result, 2, ca, in_out_params);
    lweSubMulTo(temp_result, 2, cb, in_out_params);

    //if the phase is positive, the result is 1/8
    //if the phase is positive, else the result is -1/8
    tfhe_bootstrap_FFT(result, bk->bkFFT, MU, temp_result);

    delete_LweSample(temp_result);
}


/*
 * Homomorphic bootstrapped NOT gate (doesn't need to be bootstrapped)
 * Takes in input 1 LWE samples (with message space [-1/8,1/8], noise<1/16)
 * Outputs a LWE sample (with message space [-1/8,1/8], noise<1/16)
*/
EXPORT void bootsNOT(LweSample *result, const LweSample *ca, const TFheGateBootstrappingCloudKeySet *bk) {
    const LweParams *in_out_params = bk->params->in_out_params;
    lweNegate(result, ca, in_out_params);
}


/*
 * Homomorphic bootstrapped COPY gate (doesn't need to be bootstrapped)
 * Takes in input 1 LWE samples (with message space [-1/8,1/8], noise<1/16)
 * Outputs a LWE sample (with message space [-1/8,1/8], noise<1/16)
*/
EXPORT void bootsCOPY(LweSample *result, const LweSample *ca, const TFheGateBootstrappingCloudKeySet *bk) {
    const LweParams *in_out_params = bk->params->in_out_params;
    lweCopy(result, ca, in_out_params);
}

/*
 * Homomorphic Trivial Constant gate (doesn't need to be bootstrapped)
 * Takes a boolean value)
 * Outputs a LWE sample (with message space [-1/8,1/8], noise<1/16)
*/
EXPORT void bootsCONSTANT(LweSample *result, int value, const TFheGateBootstrappingCloudKeySet *bk) {
    const LweParams *in_out_params = bk->params->in_out_params;
    static const Torus32 MU = modSwitchToTorus32(1, 8);
    lweNoiselessTrivial(result, value ? MU : -MU, in_out_params);
}


/*
 * Homomorphic bootstrapped NOR gate
 * Takes in input 2 LWE samples (with message space [-1/8,1/8], noise<1/16)
 * Outputs a LWE bootstrapped sample (with message space [-1/8,1/8], noise<1/16)
*/
EXPORT void
bootsNOR(LweSample *result, const LweSample *ca, const LweSample *cb, const TFheGateBootstrappingCloudKeySet *bk) {
    static const Torus32 MU = modSwitchToTorus32(1, 8);
    const LweParams *in_out_params = bk->params->in_out_params;

    LweSample *temp_result = new_LweSample(in_out_params);

    //compute: (0,-1/8) - ca - cb
    static const Torus32 NorConst = modSwitchToTorus32(-1, 8);
    lweNoiselessTrivial(temp_result, NorConst, in_out_params);
    lweSubTo(temp_result, ca, in_out_params);
    lweSubTo(temp_result, cb, in_out_params);

    //if the phase is positive, the result is 1/8
    //if the phase is positive, else the result is -1/8
    tfhe_bootstrap_FFT(result, bk->bkFFT, MU, temp_result);

    delete_LweSample(temp_result);
}


/*
 * Homomorphic bootstrapped AndNY Gate: not(a) and b
 * Takes in input 2 LWE samples (with message space [-1/8,1/8], noise<1/16)
 * Outputs a LWE bootstrapped sample (with message space [-1/8,1/8], noise<1/16)
*/
EXPORT void
bootsANDNY(LweSample *result, const LweSample *ca, const LweSample *cb, const TFheGateBootstrappingCloudKeySet *bk) {
    static const Torus32 MU = modSwitchToTorus32(1, 8);
    const LweParams *in_out_params = bk->params->in_out_params;

    LweSample *temp_result = new_LweSample(in_out_params);

    //compute: (0,-1/8) - ca + cb
    static const Torus32 AndNYConst = modSwitchToTorus32(-1, 8);
    lweNoiselessTrivial(temp_result, AndNYConst, in_out_params);
    lweSubTo(temp_result, ca, in_out_params);
    lweAddTo(temp_result, cb, in_out_params);

    //if the phase is positive, the result is 1/8
    //if the phase is positive, else the result is -1/8
    tfhe_bootstrap_FFT(result, bk->bkFFT, MU, temp_result);

    delete_LweSample(temp_result);
}


/*
 * Homomorphic bootstrapped AndYN Gate: a and not(b)
 * Takes in input 2 LWE samples (with message space [-1/8,1/8], noise<1/16)
 * Outputs a LWE bootstrapped sample (with message space [-1/8,1/8], noise<1/16)
*/
EXPORT void
bootsANDYN(LweSample *result, const LweSample *ca, const LweSample *cb, const TFheGateBootstrappingCloudKeySet *bk) {
    static const Torus32 MU = modSwitchToTorus32(1, 8);
    const LweParams *in_out_params = bk->params->in_out_params;

    LweSample *temp_result = new_LweSample(in_out_params);

    //compute: (0,-1/8) + ca - cb
    static const Torus32 AndYNConst = modSwitchToTorus32(-1, 8);
    lweNoiselessTrivial(temp_result, AndYNConst, in_out_params);
    lweAddTo(temp_result, ca, in_out_params);
    lweSubTo(temp_result, cb, in_out_params);

    //if the phase is positive, the result is 1/8
    //if the phase is positive, else the result is -1/8
    tfhe_bootstrap_FFT(result, bk->bkFFT, MU, temp_result);

    delete_LweSample(temp_result);
}


/*
 * Homomorphic bootstrapped OrNY Gate: not(a) or b
 * Takes in input 2 LWE samples (with message space [-1/8,1/8], noise<1/16)
 * Outputs a LWE bootstrapped sample (with message space [-1/8,1/8], noise<1/16)
*/
EXPORT void
bootsORNY(LweSample *result, const LweSample *ca, const LweSample *cb, const TFheGateBootstrappingCloudKeySet *bk) {
    static const Torus32 MU = modSwitchToTorus32(1, 8);
    const LweParams *in_out_params = bk->params->in_out_params;

    LweSample *temp_result = new_LweSample(in_out_params);

    //compute: (0,1/8) - ca + cb
    static const Torus32 OrNYConst = modSwitchToTorus32(1, 8);
    lweNoiselessTrivial(temp_result, OrNYConst, in_out_params);
    lweSubTo(temp_result, ca, in_out_params);
    lweAddTo(temp_result, cb, in_out_params);

    //if the phase is positive, the result is 1/8
    //if the phase is positive, else the result is -1/8
    tfhe_bootstrap_FFT(result, bk->bkFFT, MU, temp_result);

    delete_LweSample(temp_result);
}


/*
 * Homomorphic bootstrapped OrYN Gate: a or not(b)
 * Takes in input 2 LWE samples (with message space [-1/8,1/8], noise<1/16)
 * Outputs a LWE bootstrapped sample (with message space [-1/8,1/8], noise<1/16)
*/
EXPORT void
bootsORYN(LweSample *result, const LweSample *ca, const LweSample *cb, const TFheGateBootstrappingCloudKeySet *bk) {
    static const Torus32 MU = modSwitchToTorus32(1, 8);
    const LweParams *in_out_params = bk->params->in_out_params;

    LweSample *temp_result = new_LweSample(in_out_params);

    //compute: (0,1/8) + ca - cb
    static const Torus32 OrYNConst = modSwitchToTorus32(1, 8);
    lweNoiselessTrivial(temp_result, OrYNConst, in_out_params);
    lweAddTo(temp_result, ca, in_out_params);
    lweSubTo(temp_result, cb, in_out_params);

    //if the phase is positive, the result is 1/8
    //if the phase is positive, else the result is -1/8
    tfhe_bootstrap_FFT(result, bk->bkFFT, MU, temp_result);

    delete_LweSample(temp_result);
}




/*
 * Homomorphic bootstrapped Mux(a,b,c) = a?b:c = a*b + not(a)*c
 * Takes in input 3 LWE samples (with message space [-1/8,1/8], noise<1/16)
 * Outputs a LWE bootstrapped sample (with message space [-1/8,1/8], noise<1/16)
*/
EXPORT void bootsMUX(LweSample *result, const LweSample *a, const LweSample *b, const LweSample *c,
                     const TFheGateBootstrappingCloudKeySet *bk) {
    static const Torus32 MU = modSwitchToTorus32(1, 8);
    const LweParams *in_out_params = bk->params->in_out_params;
    const LweParams *extracted_params = &bk->params->tgsw_params->tlwe_params->extracted_lweparams;

    LweSample *temp_result = new_LweSample(in_out_params);
    LweSample *temp_result1 = new_LweSample(extracted_params);
    LweSample *u1 = new_LweSample(extracted_params);
    LweSample *u2 = new_LweSample(extracted_params);


    //compute "AND(a,b)": (0,-1/8) + a + b
    static const Torus32 AndConst = modSwitchToTorus32(-1, 8);
    lweNoiselessTrivial(temp_result, AndConst, in_out_params);
    lweAddTo(temp_result, a, in_out_params);
    lweAddTo(temp_result, b, in_out_params);
    // Bootstrap without KeySwitch
    tfhe_bootstrap_woKS_FFT(u1, bk->bkFFT, MU, temp_result);


    //compute "AND(not(a),c)": (0,-1/8) - a + c
    lweNoiselessTrivial(temp_result, AndConst, in_out_params);
    lweSubTo(temp_result, a, in_out_params);
    lweAddTo(temp_result, c, in_out_params);
    // Bootstrap without KeySwitch
    tfhe_bootstrap_woKS_FFT(u2, bk->bkFFT, MU, temp_result);

    // Add u1=u1+u2
    static const Torus32 MuxConst = modSwitchToTorus32(1, 8);
    lweNoiselessTrivial(temp_result1, MuxConst, extracted_params);
    lweAddTo(temp_result1, u1, extracted_params);
    lweAddTo(temp_result1, u2, extracted_params);
    // Key switching
    lweKeySwitch(result, bk->bkFFT->ks, temp_result1);


    delete_LweSample(u2);
    delete_LweSample(u1);
    delete_LweSample(temp_result1);
    delete_LweSample(temp_result);
}

/////new for gpu
EXPORT LweSample_16* convertBitToNumberZero(int bitSize, const TFheGateBootstrappingCloudKeySet *bk) {
    int polySize = bk->params->in_out_params->n;
    LweSample_16* temp = (LweSample_16 *)malloc(sizeof(LweSample_16));

    temp->a = (int*)calloc(bitSize*polySize, sizeof(int));
    temp->b = (int*)calloc(bitSize, sizeof(int));
    temp->current_variance = (double*)calloc(bitSize, sizeof(double));

    return temp;
}

EXPORT LweSample_16 *convertBitToNumberZero_GPU(int bitSize, const TFheGateBootstrappingCloudKeySet *bk) {
    int polySize = bk->params->in_out_params->n;
    LweSample_16 *temp = (LweSample_16 *) malloc(sizeof(LweSample_16));

    hipMalloc(&(temp->a), bitSize * polySize * sizeof(int));
    temp->b = (int *) calloc(bitSize, sizeof(int));
    //testing start
    static const Torus32 MU = modSwitchToTorus32(1, 8);
    for (int i = 0; i < bitSize; ++i) {
        temp->b[i] = -MU;
    }
    // testing end
    temp->current_variance = (double *) calloc(bitSize, sizeof(double));
    return temp;
}

EXPORT LweSample_16 *convertBitToNumberZero_GPU_2(int nOutputs, int bitSize, const TFheGateBootstrappingCloudKeySet *bk) {
    int polySize = bk->params->in_out_params->n;
    LweSample_16 *temp = (LweSample_16 *) malloc(sizeof(LweSample_16));

    hipMalloc(&(temp->a), nOutputs * bitSize * polySize * sizeof(int));
    temp->b = (int *) calloc(nOutputs * bitSize, sizeof(int));
    temp->current_variance = (double *) calloc(nOutputs * bitSize, sizeof(double));
    return temp;
}


EXPORT LweSample_16 *
newLweSample_16(int bitSize, const LweParams *params) {
    int polySize = params->n;
    LweSample_16 *temp = (LweSample_16 *) malloc(sizeof(LweSample_16));

    temp->a = (int *) calloc(bitSize * polySize, sizeof(int));
    temp->b = (int *) calloc(bitSize, sizeof(int));
    temp->current_variance = (double *) calloc(bitSize, sizeof(double));

    return temp;
}

EXPORT LweSample_16 *
newLweSample_16_2(int nOutputs, int bitSize, const LweParams *params) {
    int polySize = params->n;
    LweSample_16 *temp = (LweSample_16 *) malloc(sizeof(LweSample_16));

    temp->a = (int *) calloc(nOutputs * bitSize * polySize, sizeof(int));
    temp->b = (int *) calloc(nOutputs * bitSize, sizeof(int));
    temp->current_variance = (double *) calloc(nOutputs * bitSize, sizeof(double));

    return temp;
}

EXPORT LweSample_16* convertBitToNumber(const LweSample* input, int bitSize,
                                        const TFheGateBootstrappingCloudKeySet *bk) {
    int polySize = bk->params->in_out_params->n;

    LweSample_16* temp = (LweSample_16 *)malloc(sizeof(LweSample_16));

    temp->a = (int*)malloc(sizeof(int)*bitSize*polySize);
    temp->b = (int*)malloc(sizeof(int)*bitSize);
    temp->current_variance = (double*)malloc(sizeof(double)*bitSize);

    for (int i = 0; i < bitSize; ++i) {
        for (int j = 0; j < polySize; ++j) {
            temp->a[i * polySize + j] = (int)input[i].a[j];
        }
        temp->b[i] = input[i].b;
        temp->current_variance[i] = input[i].current_variance;
    }

    return temp;
}

EXPORT LweSample*
convertNumberToBits(LweSample_16* number, int bitSize, const TFheGateBootstrappingCloudKeySet *bk) {
    LweSample *tempCiphertext = new_gate_bootstrapping_ciphertext_array(bitSize, bk->params);
    const int n = bk->params->in_out_params->n;

    for (int i = 0; i < bitSize; ++i) {
        int startIndex = i * n;
        for (int j = 0; j < n; ++j) {
            tempCiphertext[i].a[j] = number->a[startIndex + j];
        }
        tempCiphertext[i].b = number->b[i];
        tempCiphertext[i].current_variance = number->current_variance[i];
    }
    return tempCiphertext;
}

EXPORT void
freeLweSample_16(LweSample_16* input) {
    free(input->a);
    free(input->b);
    free(input->current_variance);
    free(input);
}

int* allocateAndCopyIntVectorFromHostToDevice(int *source, int len) {
    int *d_temp;
    int bytes = len * sizeof(int);
    hipMalloc(&d_temp, bytes);
    hipMemcpy(d_temp, source, bytes, hipMemcpyHostToDevice);
    return d_temp;
}

int* allocateAndCopyIntVectorFromDeviceToHost(int *d_source, int len) {
    int bytes = len * sizeof(int);
    int *temp = (int*)malloc(bytes);
    hipMemcpy(temp, d_source, bytes, hipMemcpyDeviceToHost);
    return temp;
}

__global__ void vecAdd(int *result, int *a, int *b, int length) {
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    if (id < length) {
        result[id] = a[id] + b[id];
    }
}

__global__ void vecAddMulTo(int *result, int mulVal, int *a, int *b, int length) {
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    if (id < length) {
        result[id] = (mulVal * (a[id] + b[id]));
    }
}

void sendLweSmaple_16_a_ToGPU(LweSample_16 *sample, int bitSize, int polySize) {
    int *temp = sample->a;
    int byteLength = bitSize * polySize * sizeof(int);
    hipMalloc(&(sample->a), byteLength);
    hipMemcpy(sample->a, temp, byteLength, hipMemcpyHostToDevice);
}


EXPORT void bootsAND_16(LweSample_16 *result, const LweSample_16 *ca, const LweSample_16 *cb, int bitSize,
                        const TFheGateBootstrappingCloudKeySet *bk, hipfftDoubleComplex ****cudaBkFFT, hipfftDoubleComplex ***cudaBkFFTCoalesce,
                        Torus32 ****ks_a_gpu, Torus32 ****ks_a_gpu_extended,
                        int ***ks_b_gpu, double ***ks_cv_gpu, Torus32* ks_a_gpu_extendedPtr,
                        Torus32 *ks_b_gpu_extendedPtr, double *ks_cv_gpu_extendedPtr) {
    static const Torus32 MU = modSwitchToTorus32(1, 8);
    const LweParams *in_out_params = bk->params->in_out_params;

    int BLOCKSIZE = in_out_params->n;
    int gridSize = (int) ceil((float) (in_out_params->n * bitSize) / BLOCKSIZE);

    //compute: (0,-1/8) + ca + cb
    static const Torus32 AndConst = modSwitchToTorus32(-1, 8);
    LweSample_16 *temp_result = convertBitToNumberZero_GPU(bitSize, bk);


    for (int i = 0; i < bitSize; ++i) {
        temp_result->b[i] = AndConst;
    }

    vecAdd<<<gridSize, BLOCKSIZE>>>(temp_result->a, ca->a, cb->a, in_out_params->n * bitSize);
//    hipDeviceSynchronize();
//    cudaCheckErrors("kernel fail");

    for (int i = 0; i < bitSize; ++i) {
        temp_result->b[i] += (ca->b[i] + cb->b[i]);
//        cout << temp_result->b[i] << " ";
        temp_result->current_variance[i] += (ca->current_variance[i] + cb->current_variance[i]);
    }

    //test start
//            cout << "Inside AND:" << endl;
//    int *tempaa = new int[in_out_params->n * bitSize];
////////    int *tempba = new int[in_out_params->n * bitSize];
//    hipMemcpy(tempaa, temp_result->a, in_out_params->n * bitSize * sizeof(int), hipMemcpyDeviceToHost);
//////////    hipMemcpy(tempba, cb->a, in_out_params->n * bitSize * sizeof(int), hipMemcpyDeviceToHost);
//    for (int i = 0; i < bitSize; ++i) {
//        int sI = i * in_out_params->n;
////        cout << "ca: ";
//        for (int j = 0; j < 10; ++j) {
//            cout << tempaa[sI + j] << " ";
//        }
//        cout << endl;
////        cout << "cb: ";
////        for (int j = 0; j < 10; ++j) {
////            cout << tempba[sI + j] << " ";
////        }
//        cout << temp_result->b[i] << " ";
//        cout << endl;
//    }
//    cout << endl;
    //test end

    //if the phase is positive, the result is 1/8
    //if the phase is positive, else the result is -1/8

    tfhe_bootstrap_FFT_16(result, bk->bkFFT, MU, bitSize, temp_result, cudaBkFFT, cudaBkFFTCoalesce,
                          ks_a_gpu, ks_a_gpu_extended, ks_b_gpu, ks_cv_gpu,
                          ks_a_gpu_extendedPtr, ks_b_gpu_extendedPtr, ks_cv_gpu_extendedPtr);



//    assert(bitSize%2 == 0);

//    tfhe_bootstrap_FFT_16_2(result, bk->bkFFT, MU, 1, bitSize, temp_result, cudaBkFFT, cudaBkFFTCoalesce,
//                            ks_a_gpu, ks_a_gpu_extended, ks_b_gpu, ks_cv_gpu,
//                            ks_a_gpu_extendedPtr, ks_b_gpu_extendedPtr, ks_cv_gpu_extendedPtr);
//
//    int *temp = new int[in_out_params->n * bitSize];
//    hipMemcpy(temp, result->a, in_out_params->n * bitSize * sizeof(int), hipMemcpyDeviceToHost);
//    for (int i = 0; i < bitSize; ++i) {
//        int sI = i * in_out_params->n;
////        cout << "new: ";
//        for (int j = 0; j < 10; ++j) {
//            cout << temp[sI + j] << " ";
//        }
//        cout << endl;
////        cout << result->b[i];
////        cout << endl;
//    }

    hipFree(temp_result->a);
    temp_result->a = NULL;
    freeLweSample_16(temp_result);
}


EXPORT void bootsXOR_16(LweSample_16 *result, const LweSample_16 *ca, const LweSample_16 *cb, int bitSize,
                        const TFheGateBootstrappingCloudKeySet *bk, hipfftDoubleComplex ****cudaBkFFT, hipfftDoubleComplex ***cudaBkFFTCoalesce,
                        Torus32 ****ks_a_gpu, Torus32 ****ks_a_gpu_extended, int ***ks_b_gpu, double ***ks_cv_gpu,
                        Torus32* ks_a_gpu_extendedPtr, Torus32 *ks_b_gpu_extendedPtr, double *ks_cv_gpu_extendedPtr) {

    static const Torus32 MU = modSwitchToTorus32(1, 8);
    const LweParams *in_out_params = bk->params->in_out_params;

    int BLOCKSIZE = in_out_params->n;
    int gridSize = (int) ceil((float) (in_out_params->n * bitSize) / BLOCKSIZE);


    //compute: (0,1/4) + 2*(ca + cb)
    static const Torus32 XorConst = modSwitchToTorus32(1, 4);

    LweSample_16 *temp_result = convertBitToNumberZero_GPU(bitSize, bk);
    for (int i = 0; i < bitSize; ++i) {
        temp_result->b[i] = XorConst;
    }

    int mulVal = 2;
    vecAddMulTo<<<gridSize, BLOCKSIZE>>>(temp_result->a, mulVal, ca->a, cb->a, in_out_params->n * bitSize);
    for (int i = 0; i < bitSize; ++i) {
        temp_result->b[i] += (mulVal * (ca->b[i] + cb->b[i]));
        temp_result->current_variance[i] += ((mulVal * mulVal) * (ca->current_variance[i] + cb->current_variance[i]));
    }
    //test start
//            cout << "Inside xor: " << endl;
//    int *tempaa = new int[in_out_params->n * bitSize];
//    hipMemcpy(tempaa, temp_result->a, in_out_params->n * bitSize * sizeof(int), hipMemcpyDeviceToHost);
// //    hipMemcpy(tempba, cb->a, in_out_params->n * bitSize * sizeof(int), hipMemcpyDeviceToHost);
//    for (int i = 0; i < bitSize; ++i) {
//        int sI = i * in_out_params->n;
//        cout << "a: ";
//        for (int j = 0; j < 10; ++j) {
//            cout << tempaa[sI + j] << " ";
//        }
////        cout << temp_result->b[i] << " ";
//        cout << endl;
//    }
//    cout << endl;
//    cout << endl;
    //test end





    //if the phase is positive, the result is 1/8
    //if the phase is positive, else the result is -1/8
    tfhe_bootstrap_FFT_16(result, bk->bkFFT, MU, bitSize, temp_result, cudaBkFFT, cudaBkFFTCoalesce, ks_a_gpu,
                          ks_a_gpu_extended, ks_b_gpu, ks_cv_gpu,
                          ks_a_gpu_extendedPtr, ks_b_gpu_extendedPtr, ks_cv_gpu_extendedPtr);

//    int *temp = new int[in_out_params->n * bitSize];
//    hipMemcpy(temp, result->a, in_out_params->n * bitSize * sizeof(int), hipMemcpyDeviceToHost);
//    for (int i = 0; i < bitSize; ++i) {
//        int sI = i * in_out_params->n;
//        cout << "new: ";
//        for (int j = 0; j < 10; ++j) {
//            cout << temp[sI + j] << " ";
//        }
//        cout << result->b[i];
//        cout << endl;
//    }

    hipFree(temp_result->a);
    temp_result->a = NULL;
    freeLweSample_16(temp_result);
}

__global__ void ANDXORvecMulAllto(int *destination, int *ca, int *cb, int n, int bitSize, int length) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < length) {
        int mulVal = (id / (n * bitSize)) + 1;
        destination[id] = (mulVal * (ca[id % (n * bitSize)] + cb[id % (n * bitSize)]));
    }
}

EXPORT void bootsANDXOR_16(LweSample_16 *result, const LweSample_16 *ca, const LweSample_16 *cb, int nOutputs,
                           int bitSize, const TFheGateBootstrappingCloudKeySet *bk, hipfftDoubleComplex ****cudaBkFFT,
                           hipfftDoubleComplex ***cudaBkFFTCoalesce, Torus32 ****ks_a_gpu, Torus32 ****ks_a_gpu_extended,
                           int ***ks_b_gpu, double ***ks_cv_gpu,
                           Torus32* ks_a_gpu_extendedPtr, Torus32 *ks_b_gpu_extendedPtr, double *ks_cv_gpu_extendedPtr) {
    static const Torus32 MU = modSwitchToTorus32(1, 8);
    const LweParams *in_out_params = bk->params->in_out_params;

    //compute: (0,-1/8) + ca + cb
    static const Torus32 AndConst = modSwitchToTorus32(-1, 8);
    //compute: (0,1/4) + 2*(ca + cb)
    static const Torus32 XorConst = modSwitchToTorus32(1, 4);
    static const int mulValXor = 2;

    LweSample_16 *temp_result = convertBitToNumberZero_GPU_2(nOutputs, bitSize, bk);

    //compute temp_result->a
    int BLOCKSIZE = in_out_params->n;
    int length = in_out_params->n * bitSize * nOutputs;
    int gridSize = (int) ceil((float) (length) / BLOCKSIZE);
//    cout << "gridSize " << gridSize << endl;
    ANDXORvecMulAllto<<<gridSize, BLOCKSIZE>>>(temp_result->a, ca->a, cb->a, in_out_params->n, bitSize, length);
    //compute temp_result->b
    for (int i = 0; i < bitSize; ++i) {
        temp_result->b[i] = ca->b[i] + cb->b[i] + AndConst; //for and
        temp_result->b[i + bitSize] = mulValXor * (ca->b[i] + cb->b[i]) + XorConst;// for xor
        temp_result->current_variance[i] = ca->current_variance[i] + cb->current_variance[i]; //for and
        temp_result->current_variance[i + bitSize] = (mulValXor * mulValXor) * (ca->current_variance[i] + cb->current_variance[i]);// for xor
    }

    /*//test start
//    cout << "Inside AND:" << endl;
//    int *tempaa = new int[in_out_params->n * bitSize * nOutputs];
//    hipMemcpy(tempaa, temp_result->a, nOutputs * in_out_params->n * bitSize * sizeof(int), hipMemcpyDeviceToHost);
//    for (int i = 0; i < bitSize; ++i) {
//        int sI = i * in_out_params->n ;
//        for (int j = 0; j < 10; ++j) {
//            cout << tempaa[sI + j] << " ";
//        }
//        cout << endl;
//    }
//    cout << endl;
//    cout << "Inside XOR:" << endl;
//    for (int i = 0; i < bitSize; ++i) {
//        int sI = (bitSize + i) * in_out_params->n ;
//        for (int j = 0; j < 10; ++j) {
//            cout << tempaa[sI + j] << " ";
//        }
//        cout << endl;
//    }
//    cout << endl;*/

//    cout << "compute temp_result->b" << endl;
//    cout << "total: " << nOutputs * bitSize << endl;

    tfhe_bootstrap_FFT_16(result, bk->bkFFT, MU, bitSize * nOutputs, temp_result, cudaBkFFT, cudaBkFFTCoalesce, ks_a_gpu,
                          ks_a_gpu_extended, ks_b_gpu, ks_cv_gpu,
                          ks_a_gpu_extendedPtr, ks_b_gpu_extendedPtr, ks_cv_gpu_extendedPtr);
//    tfhe_bootstrap_FFT_16_2(result, bk->bkFFT, MU, nOutputs, bitSize, temp_result, cudaBkFFT, cudaBkFFTCoalesce,
//                            ks_a_gpu, ks_a_gpu_extended, ks_b_gpu, ks_cv_gpu,
//                            ks_a_gpu_extendedPtr, ks_b_gpu_extendedPtr, ks_cv_gpu_extendedPtr);

//    int *temp = new int[length];
//    hipMemcpy(temp, result->a, length * sizeof(int), hipMemcpyDeviceToHost);
//    for (int i = 0; i < nOutputs * bitSize; ++i) {
//        int sI = i * 500;
//        for (int j = 0; j < 10; ++j) {
//            cout << temp[sI + j] << " ";
//        }
//        cout << endl;
////        cout << result->b[i] << " " << result->current_variance[i] << endl;
//    }
//    cout << endl;
//    cout << "I am inside the function" << endl;
    hipFree(temp_result->a);
    temp_result->a = NULL;
    freeLweSample_16(temp_result);
}


__global__ void XORXORvecMulAllto(int *destination, int *ca, int *cb, int n, int bitSize, int length) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < length) {
        int mulVal = 2;
        destination[id] = (mulVal * (ca[id % (n * bitSize)] + cb[id % (n * bitSize)]));
    }
}
EXPORT void bootsXORXOR_16(LweSample_16 *result,
                           const LweSample_16 *ca1, const LweSample_16 *ca2,
                           const LweSample_16 *cb1, const LweSample_16 *cb2,
                           int nOutputs, int bitSize, const TFheGateBootstrappingCloudKeySet *bk,
                           hipfftDoubleComplex ****cudaBkFFT, hipfftDoubleComplex ***cudaBkFFTCoalesce,
                           Torus32 ****ks_a_gpu, Torus32 ****ks_a_gpu_extended, int ***ks_b_gpu, double ***ks_cv_gpu,
                           Torus32* ks_a_gpu_extendedPtr, Torus32 *ks_b_gpu_extendedPtr, double *ks_cv_gpu_extendedPtr) {

    static const Torus32 MU = modSwitchToTorus32(1, 8);
    const LweParams *in_out_params = bk->params->in_out_params;

    //compute: (0,1/4) + 2*(ca + cb)
    static const Torus32 XorConst = modSwitchToTorus32(1, 4);
    static const int mulValXor = 2, n = in_out_params->n;

    //compute temp_result->a
    int BLOCKSIZE = n;
    int length = n * bitSize;
    int gridSize = (int) ceil((float) (length) / BLOCKSIZE);
//    cout << "bitSize: " << bitSize<< endl;
//    cout << "length: " << length << endl;
//    cout << "nOut: " << nOutputs << endl;
//    cout << "gridSize: " << gridSize << endl;

    LweSample_16 *temp_result = convertBitToNumberZero_GPU_2(nOutputs, bitSize, bk);
    //compute temp_result->a
    XORXORvecMulAllto<<<gridSize, BLOCKSIZE>>>(temp_result->a, ca1->a, ca2->a, n, bitSize, length);
    XORXORvecMulAllto<<<gridSize, BLOCKSIZE>>>(temp_result->a + n, cb1->a, cb2->a, n, bitSize, length);
    //compute temp_result->b
    for (int i = 0; i < bitSize; ++i) {
        temp_result->b[i] = mulValXor * (ca1->b[i] + ca2->b[i]) + XorConst; //for and
        temp_result->b[i + bitSize] = mulValXor * (cb1->b[i] + cb2->b[i]) + XorConst;// for xor

        temp_result->current_variance[i] = (mulValXor * mulValXor) * (ca1->current_variance[i] + ca2->current_variance[i]); //for and
        temp_result->current_variance[i + bitSize] = (mulValXor * mulValXor) * (cb1->current_variance[i] + cb2->current_variance[i]);// for xor
    }

//    tfhe_bootstrap_FFT_16_2(result, bk->bkFFT, MU, nOutputs, bitSize, temp_result, cudaBkFFT, cudaBkFFTCoalesce,
//                            ks_a_gpu, ks_a_gpu_extended, ks_b_gpu, ks_cv_gpu,
//                            ks_a_gpu_extendedPtr, ks_b_gpu_extendedPtr, ks_cv_gpu_extendedPtr);
    tfhe_bootstrap_FFT_16(result, bk->bkFFT, MU, bitSize * nOutputs, temp_result, cudaBkFFT, cudaBkFFTCoalesce, ks_a_gpu,
                          ks_a_gpu_extended, ks_b_gpu, ks_cv_gpu,
                          ks_a_gpu_extendedPtr, ks_b_gpu_extendedPtr, ks_cv_gpu_extendedPtr);

    hipFree(temp_result->a);
    temp_result->a = NULL;
    freeLweSample_16(temp_result);
}


__global__ void XORXORvecMulAllto_vector(int *destination, int *ca, int *cb, int n, int bitSize, int length) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < length) {
        int mulVal = 2;
        destination[id] = (mulVal * (ca[id] + cb[id]));
    }
}

EXPORT void bootsXORXOR_16_vector(LweSample_16 *result,
                           const LweSample_16 *ca1, const LweSample_16 *ca2,
                           const LweSample_16 *cb1, const LweSample_16 *cb2,
                           int vLength, int nOutputs, int bitSize, const TFheGateBootstrappingCloudKeySet *bk,
                           hipfftDoubleComplex ****cudaBkFFT, hipfftDoubleComplex ***cudaBkFFTCoalesce,
                           Torus32 ****ks_a_gpu, Torus32 ****ks_a_gpu_extended, int ***ks_b_gpu, double ***ks_cv_gpu,
                           Torus32* ks_a_gpu_extendedPtr, Torus32 *ks_b_gpu_extendedPtr, double *ks_cv_gpu_extendedPtr) {

    static const Torus32 MU = modSwitchToTorus32(1, 8);
    const LweParams *in_out_params = bk->params->in_out_params;

    //compute: (0,1/4) + 2*(ca + cb)
    static const Torus32 XorConst = modSwitchToTorus32(1, 4);
    static const int mulValXor = 2, n = in_out_params->n;

    int totalBitSize = vLength * bitSize;
    //compute temp_result->a
    int BLOCKSIZE = n;
    int length = n * totalBitSize;//svLength * bitSize;
    int gridSize = (int) ceil((float) (length) / BLOCKSIZE);
//    cout << "vLen: " << vLength << endl;
//    cout << "bitSize: " << bitSize<< endl;
//    cout << "length: " << length << endl;
//    cout << "nOut: " << nOutputs << endl;
//    cout << "gridSize: " << gridSize << endl;

    LweSample_16 *temp_result = convertBitToNumberZero_GPU_2(nOutputs * vLength, bitSize, bk);
    //compute temp_result->a
    XORXORvecMulAllto_vector<<<gridSize, BLOCKSIZE>>>(temp_result->a, ca1->a, ca2->a, n, bitSize, length);
    XORXORvecMulAllto_vector<<<gridSize, BLOCKSIZE>>>(temp_result->a + n * vLength, cb1->a, cb2->a, n, bitSize, length);
    //compute temp_result->b
    for (int i = 0; i < totalBitSize; ++i) {
        temp_result->b[i] = mulValXor * (ca1->b[i] + ca2->b[i]) + XorConst; //for and
        temp_result->b[i + totalBitSize] = mulValXor * (cb1->b[i] + cb2->b[i]) + XorConst;// for xor

        temp_result->current_variance[i] = (mulValXor * mulValXor) * (ca1->current_variance[i] + ca2->current_variance[i]); //for and
        temp_result->current_variance[i + totalBitSize] = (mulValXor * mulValXor) * (cb1->current_variance[i] + cb2->current_variance[i]);// for xor
    }
//    cout << "HEREZZZZZZZZ----" << endl;

//    tfhe_bootstrap_FFT_16_2(result, bk->bkFFT, MU, nOutputs, bitSize * vLength, temp_result, cudaBkFFT, cudaBkFFTCoalesce,
//                            ks_a_gpu, ks_a_gpu_extended, ks_b_gpu, ks_cv_gpu,
//                            ks_a_gpu_extendedPtr, ks_b_gpu_extendedPtr, ks_cv_gpu_extendedPtr);
//
    tfhe_bootstrap_FFT_16(result, bk->bkFFT, MU, bitSize * nOutputs * vLength, temp_result, cudaBkFFT, cudaBkFFTCoalesce, ks_a_gpu,
                          ks_a_gpu_extended, ks_b_gpu, ks_cv_gpu,
                          ks_a_gpu_extendedPtr, ks_b_gpu_extendedPtr, ks_cv_gpu_extendedPtr);

//    tfhe_bootstrap_FFT_16_2_vector(result, bk->bkFFT, MU, vLength, nOutputs, bitSize, temp_result, cudaBkFFT,
//                                   cudaBkFFTCoalesce, ks_a_gpu, ks_a_gpu_extended, ks_b_gpu, ks_cv_gpu,
//                                   ks_a_gpu_extendedPtr, ks_b_gpu_extendedPtr, ks_cv_gpu_extendedPtr);

    hipFree(temp_result->a);
    temp_result->a = NULL;
    freeLweSample_16(temp_result);
}




__global__ void ANDXORvecMulAllto_vector(int *destination, int *ca, int *cb, int vLength, int bitSize, int n, int length) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < length) {
        int mulVal = (id / (vLength * bitSize * n)) + 1;
        destination[id] = (mulVal * (ca[id % (vLength * bitSize * n)] + cb[id % (vLength * bitSize * n)]));
    }
}

EXPORT void bootsANDXOR_16_vector(LweSample_16 *result, const LweSample_16 *ca, const LweSample_16 *cb, int nOutputs,
                                  int vLength, int bitSize, const TFheGateBootstrappingCloudKeySet *bk,
                                  hipfftDoubleComplex ****cudaBkFFT, hipfftDoubleComplex ***cudaBkFFTCoalesce,
                                  Torus32 ****ks_a_gpu, Torus32 ****ks_a_gpu_extended,
                                  int ***ks_b_gpu, double ***ks_cv_gpu,
                                  Torus32 *ks_a_gpu_extendedPtr, Torus32 *ks_b_gpu_extendedPtr,
                                  double *ks_cv_gpu_extendedPtr) {

    static const Torus32 MU = modSwitchToTorus32(1, 8);
    const LweParams *in_out_params = bk->params->in_out_params;
    const int n = in_out_params->n;//500

    //compute: (0,-1/8) + ca + cb
    static const Torus32 AndConst = modSwitchToTorus32(-1, 8);
    //compute: (0,1/4) + 2*(ca + cb)
    static const Torus32 XorConst = modSwitchToTorus32(1, 4);
    static const int mulValXor = 2;

    LweSample_16 *temp_result = convertBitToNumberZero_GPU_2(nOutputs, vLength * bitSize, bk);
    int BLOCKSIZE = 1024;
    int length = vLength * bitSize * nOutputs * n;
    int gridSize = (int) ceil((float) (length) / BLOCKSIZE);
    ANDXORvecMulAllto_vector<<<gridSize, BLOCKSIZE>>>(temp_result->a, ca->a, cb->a, vLength, bitSize, n, length);

    //compute temp_result->b
    int totalBitSize = vLength * bitSize;
    for (int i = 0; i < totalBitSize; ++i) {
        temp_result->b[i] = ca->b[i] + cb->b[i] + AndConst; //for and
        temp_result->b[i + totalBitSize] = mulValXor * (ca->b[i] + cb->b[i]) + XorConst;// for xor
        temp_result->current_variance[i] = ca->current_variance[i] + cb->current_variance[i]; //for and
        temp_result->current_variance[i + totalBitSize] = (mulValXor * mulValXor) * (ca->current_variance[i] + cb->current_variance[i]);// for xor
    }

    //test start
//    cout << "Inside AND:" << endl;
//    int *tempaa = new int[n * bitSize * nOutputs * vLength];
//    hipMemcpy(tempaa, temp_result->a, vLength * nOutputs * n * bitSize * sizeof(int), hipMemcpyDeviceToHost);
//    for (int i = 0; i < bitSize * vLength; ++i) {
//        int sI = i * n ;
//        for (int j = 0; j < 10; ++j) {
//            cout << tempaa[sI + j] << " ";
//        }
//        cout << endl;
//    }
//    cout << endl;
//    cout << "Inside XOR:" << endl;
//    for (int i = 0; i < bitSize * vLength; ++i) {
//        int sI = (bitSize * vLength + i) * n ;
//        for (int j = 0; j < 10; ++j) {
//            cout << tempaa[sI + j] << " ";
//        }
//        cout << endl;
//    }
//    cout << endl;


//    cout << "HEREZZZZZZZZZZZ" << endl;

    tfhe_bootstrap_FFT_16(result, bk->bkFFT, MU, nOutputs * vLength * bitSize, temp_result, cudaBkFFT, cudaBkFFTCoalesce,
                          ks_a_gpu, ks_a_gpu_extended, ks_b_gpu, ks_cv_gpu,
                          ks_a_gpu_extendedPtr, ks_b_gpu_extendedPtr, ks_cv_gpu_extendedPtr);
//
//    tfhe_bootstrap_FFT_16_2(result, bk->bkFFT, MU, nOutputs, vLength * bitSize, temp_result, cudaBkFFT, cudaBkFFTCoalesce,
//                            ks_a_gpu, ks_a_gpu_extended, ks_b_gpu, ks_cv_gpu,
//                            ks_a_gpu_extendedPtr, ks_b_gpu_extendedPtr, ks_cv_gpu_extendedPtr);

//    cout << "HEREZZZZZZZZZZZ--" << endl;

//    tfhe_bootstrap_FFT_16_2_vector(result, bk->bkFFT, MU, vLength, nOutputs, bitSize, temp_result, cudaBkFFT,
//                                   cudaBkFFTCoalesce, ks_a_gpu, ks_a_gpu_extended, ks_b_gpu, ks_cv_gpu,
//                                   ks_a_gpu_extendedPtr, ks_b_gpu_extendedPtr, ks_cv_gpu_extendedPtr);

//    int *temp = new int[length];
//    hipMemcpy(temp, result->a, length * sizeof(int), hipMemcpyDeviceToHost);
//    cout << "AND PART" << endl;
//    for (int i = 0; i < 16 * bitSize; ++i) {
//        int sI = i * 500;
//        for (int j = 0; j < 10; ++j) {
//            cout << temp[sI + j] << " ";
//        }
//        cout << endl;
////        cout << result->b[i] << " " << result->current_variance[i] << endl;
//    }
//    cout << endl;
//    cout << endl << "XOR PART" << endl;
//    for (int i = 0; i < 16 * bitSize; ++i) {
//        int sIB = bitSize * vLength ;
//        int sI = i * 500 + bitSize * vLength * 500;
//        for (int j = 0; j < 10; ++j) {
//            cout << temp[sI + j] << " ";
//        }
//        cout << endl;
////        cout << result->b[sIB + i] << " " << result->current_variance[sI + i] << endl;
//    }
//    cout << endl;



    hipFree(temp_result->a);
    temp_result->a = NULL;
    freeLweSample_16(temp_result);
}












































//only used for multiplication
__global__ void vecAdd_MULT(int *result, int *a, int *b, int bAStart, int n, int length) {
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    if (id < length) {
        result[id] = a[id] + b[(id % n) + bAStart];
    }
}



EXPORT void bootsAND_MULT(LweSample_16 *result,
                          const LweSample_16 *ca, const LweSample_16 *cb,
                          int resBitSize, int bitSize_A, int bIndex,
                          const TFheGateBootstrappingCloudKeySet *bk, hipfftDoubleComplex ****cudaBkFFT,
                          hipfftDoubleComplex ***cudaBkFFTCoalesce, Torus32 ****ks_a_gpu, Torus32 ****ks_a_gpu_extended,
                          int ***ks_b_gpu, double ***ks_cv_gpu,
                          Torus32 *ks_a_gpu_extendedPtr, Torus32 *ks_b_gpu_extendedPtr, double *ks_cv_gpu_extendedPtr) {

    assert(bitSize_A == resBitSize);
    static const Torus32 MU = modSwitchToTorus32(1, 8);
    const LweParams *in_out_params = bk->params->in_out_params;

    int n = in_out_params->n;
    int BLOCKSIZE = 1024;

    int gridSize = (int) ceil((float) (in_out_params->n * bitSize_A) / BLOCKSIZE);
    int bAstartIndex = bIndex * n;

    //compute: (0,-1/8) + ca + cb
    static const Torus32 AndConst = modSwitchToTorus32(-1, 8);
    LweSample_16 *temp_result = convertBitToNumberZero_GPU(bitSize_A, bk);

    for (int i = 0; i < bitSize_A; ++i) {
        temp_result->b[i] = AndConst;
    }

    vecAdd_MULT<<<gridSize, BLOCKSIZE>>>(temp_result->a, ca->a, cb->a, bAstartIndex, n, n * bitSize_A);

    for (int i = 0; i < bitSize_A; ++i) {
        temp_result->b[i] += (ca->b[i] + cb->b[bIndex]);
        temp_result->current_variance[i] += (ca->current_variance[i] + cb->current_variance[bIndex]);
    }

    int bitSize = bitSize_A;
    tfhe_bootstrap_FFT_16(result, bk->bkFFT, MU, bitSize, temp_result, cudaBkFFT, cudaBkFFTCoalesce,
                          ks_a_gpu, ks_a_gpu_extended, ks_b_gpu, ks_cv_gpu,
                          ks_a_gpu_extendedPtr, ks_b_gpu_extendedPtr, ks_cv_gpu_extendedPtr);

    //dhor tokta mar perek
    //find out later on
//    hipMemset(result->a + (n * bitSize_A), 0, n * (resBitSize - bitSize_A) * sizeof(int));
//    for (int i = bitSize_A; i < resBitSize; ++i) {
//        cout << result->b[i] << " ";
//    }
//    cout << endl;

    hipFree(temp_result->a);
    temp_result->a = NULL;
    freeLweSample_16(temp_result);
}


EXPORT void bootsAND_MULT_con(LweSample_16 *result,
                              LweSample_16 **ca, LweSample_16 **cb,
                              int nConMul, int resBitSize, int bitSize_A, int bIndex,
                              const TFheGateBootstrappingCloudKeySet *bk, hipfftDoubleComplex ****cudaBkFFT,
                              hipfftDoubleComplex ***cudaBkFFTCoalesce, Torus32 ****ks_a_gpu,
                              Torus32 ****ks_a_gpu_extended, int ***ks_b_gpu, double ***ks_cv_gpu,
                              Torus32 *ks_a_gpu_extendedPtr, Torus32 *ks_b_gpu_extendedPtr,
                              double *ks_cv_gpu_extendedPtr) {

    assert(bitSize_A == resBitSize);
    static const Torus32 MU = modSwitchToTorus32(1, 8);
    const LweParams *in_out_params = bk->params->in_out_params;

    int n = in_out_params->n;
    int BLOCKSIZE = n;

    int gridSize = (int) ceil((float) (n * bitSize_A) / BLOCKSIZE);
    int bAstartIndex = bIndex * n;

    //compute: (0,-1/8) + ca + cb
    static const Torus32 AndConst = modSwitchToTorus32(-1, 8);
    LweSample_16 *temp_result = convertBitToNumberZero_GPU(bitSize_A * nConMul, bk);

//    for (int i = 0; i < bitSize_A; ++i) {
//        temp_result->b[i] = AndConst;
//    }

    for (int i = 0; i < nConMul; ++i) {
        vecAdd_MULT<<<gridSize, BLOCKSIZE>>>(temp_result->a + i * bitSize_A * n, ca[i]->a, cb[i]->a, bAstartIndex, n, n * bitSize_A);
    }
    for (int j = 0; j < nConMul; ++j) {
        int sI = j * bitSize_A;
        for (int i = 0; i < bitSize_A; ++i) {
            int sI2 = sI + i;
            temp_result->b[sI2] = (ca[j]->b[i] + cb[j]->b[bIndex]) + AndConst;
            temp_result->current_variance[sI2] = (ca[j]->b[i] + cb[j]->b[bIndex]);
        }
    }

    int toalBitSize = bitSize_A * nConMul;
//    cout << "totalBitSize:" << toalBitSize << endl;
//    int nOutputs = 2;
//    int vLength = nConMul/2;
//    int bitSize = bitSize_A;
    tfhe_bootstrap_FFT_16(result, bk->bkFFT, MU, toalBitSize, temp_result, cudaBkFFT, cudaBkFFTCoalesce,
                          ks_a_gpu, ks_a_gpu_extended, ks_b_gpu, ks_cv_gpu,
                          ks_a_gpu_extendedPtr, ks_b_gpu_extendedPtr, ks_cv_gpu_extendedPtr);
//        cout << "bootsAND_MULT_con: bitSize: " << bitSize << " vLen: " << vLength << endl;
//        if (nConMul % 2 == 1) {
//            cout << "ERROR: Provide even number of vector" << endl;
//            exit(1);
//        }
//    tfhe_bootstrap_FFT_16_2_vector(result, bk->bkFFT, MU, vLength, nOutputs, bitSize, temp_result, cudaBkFFT,
//                                   cudaBkFFTCoalesce, ks_a_gpu, ks_a_gpu_extended, ks_b_gpu, ks_cv_gpu,
//                                   ks_a_gpu_extendedPtr, ks_b_gpu_extendedPtr, ks_cv_gpu_extendedPtr);

    hipFree(temp_result->a);
    temp_result->a = NULL;
    freeLweSample_16(temp_result);
}






















//(a xor b) and c

EXPORT void bootsXOR_AND(LweSample *result, const LweSample *ca, const LweSample *cb, const LweSample *cc, const TFheGateBootstrappingCloudKeySet *bk) {

    static const Torus32 MU = modSwitchToTorus32(1, 8);
    const LweParams *in_out_params = bk->params->in_out_params;

    LweSample *temp_result = new_LweSample(in_out_params);
    LweSample *temp_result1 = new_LweSample(in_out_params);
    LweSample *temp_result2 = new_LweSample(in_out_params);

    //compute: (0,1/4) + 2*(ca + cb)
    static const Torus32 XorConst = modSwitchToTorus32(1, 4);
    static const Torus32 AndConst = modSwitchToTorus32(-1, 8);

    lweNoiselessTrivial(temp_result, XorConst, in_out_params);

    lweAddMulTo(temp_result, 2, ca, in_out_params);
    lweAddMulTo(temp_result, 2, cb, in_out_params);

    tfhe_bootstrap_FFT(result, bk->bkFFT, MU, temp_result);


    lweNoiselessTrivial(temp_result2, AndConst, in_out_params);

    lweAddTo(temp_result2, cc, in_out_params);
    lweAddTo(temp_result2, result, in_out_params);

//    static const Torus32 MU = modSwitchToTorus32(1, 8);
//    const LweParams *in_out_params = bk->params->in_out_params;

//    LweSample *temp_result = new_LweSample(in_out_params);

    //compute: (0,-1/8) + ca + cb


//    lweAddTo(temp_result, cb, in_out_params);

    tfhe_bootstrap_FFT(result, bk->bkFFT, MU, temp_result2);

    delete_LweSample(temp_result);
}

__global__ void reverseLweSample(int *dest, int *source, int length) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < length) {
        dest[id] = -source[id];
    }
}

void bootsNOT_16(LweSample_16 *output, LweSample_16 *input, int bitSize, int params_n) {
    int length = bitSize * params_n, BLOCKSIZE = 1024, gridSize = (int) ceil((float) (length) / BLOCKSIZE);
    reverseLweSample<<<gridSize, BLOCKSIZE>>>(output->a, input->a, length);
    for (int i = 0; i < bitSize; ++i) {
        output->b[i] = -input->b[i];
        output->current_variance[i] = input->current_variance[i];
    }
}







//add vector




__global__ void ANDvec_vector(int *destination, int *ca, int *cb, int vLength, int bitSize, int n, int length) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < length) {
        destination[id] = ca[id] + cb[id];
    }
}

EXPORT void bootsAND_16_vector(LweSample_16 *result, const LweSample_16 *ca, const LweSample_16 *cb, int nOutputs,
                                  int vLength, int bitSize, const TFheGateBootstrappingCloudKeySet *bk,
                                  hipfftDoubleComplex ****cudaBkFFT, hipfftDoubleComplex ***cudaBkFFTCoalesce,
                                  Torus32 ****ks_a_gpu, Torus32 ****ks_a_gpu_extended,
                                  int ***ks_b_gpu, double ***ks_cv_gpu,
                                  Torus32 *ks_a_gpu_extendedPtr, Torus32 *ks_b_gpu_extendedPtr,
                                  double *ks_cv_gpu_extendedPtr) {
    assert(nOutputs == 1);

    static const Torus32 MU = modSwitchToTorus32(1, 8);
    const LweParams *in_out_params = bk->params->in_out_params;
    const int n = in_out_params->n;//500

    //compute: (0,-1/8) + ca + cb
    static const Torus32 AndConst = modSwitchToTorus32(-1, 8);


    LweSample_16 *temp_result = convertBitToNumberZero_GPU_2(nOutputs, vLength * bitSize, bk);
    int BLOCKSIZE = 1024;
    int length = vLength * bitSize * nOutputs * n;
    int gridSize = (int) ceil((float) (length) / BLOCKSIZE);
    ANDvec_vector<<<gridSize, BLOCKSIZE>>>(temp_result->a, ca->a, cb->a, vLength, bitSize, n, length);

    //compute temp_result->b
    int totalBitSize = vLength * bitSize;
    for (int i = 0; i < totalBitSize; ++i) {
        temp_result->b[i] = ca->b[i] + cb->b[i] + AndConst; //for and
        temp_result->current_variance[i] = ca->current_variance[i] + cb->current_variance[i]; //for and
    }



//    cout << "xxxxxxxxxxxxxxxxxxxx" << endl;
//    cout << nOutputs << endl;
    tfhe_bootstrap_FFT_16(result, bk->bkFFT, MU, vLength * bitSize * nOutputs, temp_result, cudaBkFFT, cudaBkFFTCoalesce,
                          ks_a_gpu, ks_a_gpu_extended, ks_b_gpu, ks_cv_gpu,
                          ks_a_gpu_extendedPtr, ks_b_gpu_extendedPtr, ks_cv_gpu_extendedPtr);

//    if (vLength % 2 == 1 && vLength < 2) {
////        cout <<  "vLen: " << vLength << " bitSize: " << bitSize << endl;
//        tfhe_bootstrap_FFT_16(result, bk->bkFFT, MU, bitSize, temp_result, cudaBkFFT, cudaBkFFTCoalesce,
//                              ks_a_gpu, ks_a_gpu_extended, ks_b_gpu, ks_cv_gpu,
//                              ks_a_gpu_extendedPtr, ks_b_gpu_extendedPtr, ks_cv_gpu_extendedPtr);
////        bitSize = bitSize/2;
////        tfhe_bootstrap_FFT_16_2_vector(result, bk->bkFFT, MU, 2, 2, bitSize/4, temp_result, cudaBkFFT,
////                                       cudaBkFFTCoalesce, ks_a_gpu, ks_a_gpu_extended, ks_b_gpu, ks_cv_gpu,
////                                       ks_a_gpu_extendedPtr, ks_b_gpu_extendedPtr, ks_cv_gpu_extendedPtr);
//    } else {
//        nOutputs = 2;
//        vLength = vLength / 2;
//        tfhe_bootstrap_FFT_16_2_vector(result, bk->bkFFT, MU, vLength, nOutputs, bitSize, temp_result, cudaBkFFT,
//                                       cudaBkFFTCoalesce, ks_a_gpu, ks_a_gpu_extended, ks_b_gpu, ks_cv_gpu,
//                                       ks_a_gpu_extendedPtr, ks_b_gpu_extendedPtr, ks_cv_gpu_extendedPtr);
//    }



    hipFree(temp_result->a);
    temp_result->a = NULL;
    freeLweSample_16(temp_result);
}




__global__ void SUBvec_vector(int *destination, int *ca, int *cc, int vLength, int bitSize, int n, int length) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < length) {
        destination[id] = cc[id] - ca[id];
    }
}


EXPORT void bootsMUX_16_vector(LweSample_16 *result, const LweSample_16 *ca, const LweSample_16 *cb,
                               const LweSample_16 *cc, int vLength, int bitSize, const TFheGateBootstrappingCloudKeySet *bk,
                               hipfftDoubleComplex ****cudaBkFFT, hipfftDoubleComplex ***cudaBkFFTCoalesce,
                               Torus32 ****ks_a_gpu, Torus32 ****ks_a_gpu_extended,
                               int ***ks_b_gpu, double ***ks_cv_gpu,
                               Torus32 *ks_a_gpu_extendedPtr, Torus32 *ks_b_gpu_extendedPtr,
                               double *ks_cv_gpu_extendedPtr) {

    static const Torus32 MU = modSwitchToTorus32(1, 8);
    const LweParams *in_out_params = bk->params->in_out_params;
    const LweParams *extracted_params = &bk->params->tgsw_params->tlwe_params->extracted_lweparams;
    const int n = in_out_params->n;//500
    const int extracted_n = extracted_params->n;//1024
    int nOutputs = 2;
    //for now vLength = 1
    assert(vLength == 1);
//    cout << "n: " << n << endl;
//    cout << "nOutputs: " << nOutputs << endl;
//    cout << "vLength: " << vLength << endl;
//    cout << "extracted_n: " << extracted_n << endl;

    int ex_length = vLength * bitSize * extracted_n;//ex_length does not include nOutputs
    int length = vLength * bitSize * n;//length does not include nOutputs
    int BLOCKSIZE = 1024;

    LweSample_16 *temp_result = convertBitToNumberZero_GPU_2(nOutputs, vLength * bitSize, bk);
    LweSample_16 *u = newLweSample_16_2(nOutputs, vLength * bitSize, extracted_params);
    LweSample_16 *ex_temp_result = newLweSample_16_2(1, vLength * bitSize, extracted_params);
    free(u->a);
    free(ex_temp_result->a);
    hipMalloc(&(u->a), ex_length * nOutputs * sizeof(int));
    hipMalloc(&(ex_temp_result->a), ex_length * sizeof(int));

    //compute: (0,-1/8) + ca + cb
    static const Torus32 AndConst = modSwitchToTorus32(-1, 8);
    static const Torus32 MuxConst = modSwitchToTorus32(1, 8);

    int gridSize = (int) ceil((float) (length) / BLOCKSIZE);
    ANDvec_vector<<<gridSize, BLOCKSIZE>>>(temp_result->a, ca->a, cb->a, vLength, bitSize, n, length);
    SUBvec_vector<<<gridSize, BLOCKSIZE>>>(temp_result->a + length, ca->a, cc->a, vLength, bitSize, n, length);

    //compute temp_result->b
    int totalBitSize = vLength * bitSize;
    for (int i = 0; i < totalBitSize; ++i) {
        temp_result->b[i] = ca->b[i] + cb->b[i] + AndConst;
        temp_result->current_variance[i] = ca->current_variance[i] + cb->current_variance[i];

        temp_result->b[i + totalBitSize] = - ca->b[i] + cc->b[i] + AndConst;
        temp_result->current_variance[i + totalBitSize] = - ca->current_variance[i] + cc->current_variance[i]; //for and
    }

    tfhe_bootstrap_woKS_FFT_16_2_vector(u, bk->bkFFT, MU, vLength, nOutputs, bitSize, temp_result, cudaBkFFT, cudaBkFFTCoalesce);

    gridSize = (int) ceil((float) (ex_length) / BLOCKSIZE);
    ANDvec_vector<<<gridSize, BLOCKSIZE>>>(ex_temp_result->a, u->a, u->a + ex_length,
                                            vLength, bitSize, extracted_n, ex_length);

    for (int i = 0; i < vLength * bitSize; ++i) {
        ex_temp_result->b[i] = u->b[i] + u->b[i + vLength * bitSize] + MuxConst;
        ex_temp_result->current_variance[i] = u->current_variance[i] + u->current_variance[i + vLength * bitSize];
    }

//    lweKeySwitch_16(result, bk->ks, u, bitSize, ks_a_gpu, ks_a_gpu_extended, ks_b_gpu, ks_cv_gpu,
//                    ks_a_gpu_extendedPtr, ks_b_gpu_extendedPtr, ks_cv_gpu_extendedPtr);

    lweKeySwitch_16_2_vector(result, bk->bkFFT->ks, ex_temp_result, vLength, 1, bitSize, ks_a_gpu, ks_a_gpu_extended, ks_b_gpu,
                             ks_cv_gpu, ks_a_gpu_extendedPtr, ks_b_gpu_extendedPtr, ks_cv_gpu_extendedPtr);

//    tfhe_bootstrap_FFT_16_2_vector(result, bk->bkFFT, MU, vLength, nOutputs, bitSize, temp_result, cudaBkFFT,
//                                   cudaBkFFTCoalesce, ks_a_gpu, ks_a_gpu_extended, ks_b_gpu, ks_cv_gpu,
//                                   ks_a_gpu_extendedPtr, ks_b_gpu_extendedPtr, ks_cv_gpu_extendedPtr);


//    if (vLength % 2 == 1 && vLength < 2) {
//        cout <<  "Odd number in bootsAND_16_vector" << endl;
//    }
//    nOutputs = 2;
//    vLength = vLength/2;
//    tfhe_bootstrap_FFT_16_2_vector(result, bk->bkFFT, MU, vLength, nOutputs, bitSize, temp_result, cudaBkFFT,
//                                   cudaBkFFTCoalesce, ks_a_gpu, ks_a_gpu_extended, ks_b_gpu, ks_cv_gpu,
//                                   ks_a_gpu_extendedPtr, ks_b_gpu_extendedPtr, ks_cv_gpu_extendedPtr);
//
//
//
//    hipFree(temp_result->a);
//    temp_result->a = NULL;
//    freeLweSample_16(temp_result);
}


__device__ int modSwitchFromTorus32_GPU_device(Torus32 phase, int Msize){
    uint64_t interv = ((UINT64_C(1)<<63)/Msize)*2; // width of each intervall
    uint64_t half_interval = interv/2; // begin of the first intervall
    uint64_t phase64 = (uint64_t(phase)<<32) + half_interval;
    //floor to the nearest multiples of interv
    return phase64/interv;
}



__global__ void bootstrappingUptoBlindRotate_OneBit(int *accum_a_b, int *temp_accum_a_b, int *bara_g, Torus32 MU, int *temp_res_a, int temp_res_b, double temp_res_cv,
                                      hipfftDoubleComplex *cudaBkFFTCoalesceExt) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < 1024) {
        //x is temp_res
        register int n = 500, N = 1024, _2N = 2048, Ns2 = 512, Nx2 = 2048;
        //tfhe_bootstrap_FFT_16--> u
//        __shared__ int u_a[1024], u_b;//N
//        __shared__ double u_cv;
        //tfhe_bootstrap_woKS_FFT_16
//        __shared__ int bara[1024];//N//torusPolyTestvect_coef[1024],
        register int barb;

        bara_g[id] = 0;
//        torusPolyTestvect_coef[id] = MU;
        if (id < n) {//500
            bara_g[id] = modSwitchFromTorus32_GPU_device(temp_res_a[id], Nx2);
        }
        __syncthreads();

        barb = modSwitchFromTorus32_GPU_device(temp_res_b, Nx2);
        //tfhe_blindRotateAndExtract_FFT_16 -> here v = torusPolyTestvect_coef
        __shared__ int testvectbis[1024];//N
        //torusPolynomialMulByXai_16 -> res ->testvectbis, v-> torusPolyTestvect_coef
        register int a = _2N - barb;

        if (a < N) {//1024
            if (id < a) {
                testvectbis[id] = -MU;//torusPolyTestvect_coef[id - a + N];
            } else {
                testvectbis[id] = MU;//torusPolyTestvect_coef[id - a];
            }
        } else {
            register int aa = a - N;
            if (id < aa) {
                testvectbis[id] = MU;//torusPolyTestvect_coef[id - aa + N];
            } else {
                testvectbis[id] = -MU;//torusPolyTestvect_coef[id - aa];
            }
        }
        __syncthreads();
        accum_a_b[id] = 0;//accum_a
        accum_a_b[1024 + id] = testvectbis[id];

        temp_accum_a_b[id] = 0;//accum_a
        temp_accum_a_b[1024 + id] = 0;

//        bara_g[id] = bara[id];
    }
}

__global__ void prepareForiFFT_1_Bit(int *des, int *decaCoalesce, hipfftDoubleReal *d_rev_in,
                                                     int *bara, int baraIndex, int *source) {
    register int id = blockIdx.x * blockDim.x + threadIdx.x;
    register int N = 1024, _2N = 2048, Ns2 = 512;

    register int tIndex = id % N;
    register int a = bara[baraIndex];
    register int aa = a - N;

    register bool l1 = a < N, l2 = tIndex < a, l3 = tIndex < aa;

    int des_id = l1 * (l2 * (-source[id - a + N] - source[id]) + (!l2) * (source[id - a] - source[id]))
                  + (!l1) * (l3 * (source[id - aa + N] - source[id])
                             + (!l3) * (-source[id - aa] - source[id]));

    register uint32_t halfBg = 512, maskMod = 1023, Bgbit = 10;
//    register uint32_t offset = 2149580800;


    register int p = 0;
    register int decal = (32 - (p + 1) * Bgbit);
    register uint32_t temp1 = (((uint32_t)(des_id + 2149580800)) >> decal) & maskMod;//offset

    register int xxxxx1 = (temp1 - halfBg);
//    decaCoalesce[((id / (N)) * (N)) + id] =
//            (middleBlock) * xxxxx1 + (!middleBlock) * (decaCoalesce[((id / (N)) * (N)) + id]);

    p = 1;
    decal = (32 - (p + 1) * Bgbit);
    temp1 = (((uint32_t)(des_id + 2149580800)) >> decal) & maskMod;//offset
    register int xxxxx2 = temp1 - halfBg;

//    decaCoalesce[((id / (N)) * (N)) + id + (N)] = middleBlock * xxxxx2 + (!middleBlock) * decaCoalesce[((id / (N)) * (N)) + id + (N)];


    register int bIndex = id / N;

    int destTod_rev_in = bIndex * _2N + tIndex + (bIndex >= 1) * N * 2;

    d_rev_in[destTod_rev_in] = xxxxx1/2.;
    d_rev_in[destTod_rev_in + 1024] = -xxxxx1/2.;

    destTod_rev_in += 2 * 1024;
    d_rev_in[destTod_rev_in] = xxxxx2/2.;
    d_rev_in[destTod_rev_in + 1024] = -xxxxx2/2.;
}


__global__  void prepareForFFT_1_Bit(hipfftDoubleComplex *cuDecaFFTCoalesce, hipfftDoubleComplex *tmpa_gpuCoal,
                                                 hipfftDoubleComplex *d_in, hipfftDoubleComplex *d_rev_out,
                                                 hipfftDoubleComplex *bki,  int keyIndex,
                                                 int N, int Ns2, int length) {

    register int id = blockIdx.x*blockDim.x+threadIdx.x;
    register int k = 1, kpl = 4, keySI = keyIndex * (k + 1) * kpl * Ns2, aID, bID, offset;



//    if(id < 512) {

        int tempId = id;
        int bitIndex = tempId / Ns2;
        register hipfftDoubleComplex v0 = d_rev_out[2 * tempId + 1 + bitIndex];//d_rev_out[2 * id + 1 + bitIndex];
//        cuDecaFFTCoalesce[tempId] = v0;

        tempId = tempId + (Ns2);
        bitIndex = (tempId) / Ns2;
        register hipfftDoubleComplex v1 = d_rev_out[2 * tempId + 1 + bitIndex];
//        cuDecaFFTCoalesce[tempId] = v1;

        tempId = tempId + (Ns2);
        bitIndex = (tempId) / Ns2;
        register hipfftDoubleComplex v2 = d_rev_out[2 * tempId + 1 + bitIndex];
//        cuDecaFFTCoalesce[tempId] = v2;

        tempId = tempId + (Ns2);
        bitIndex = (tempId) / Ns2;
        register hipfftDoubleComplex v3 = d_rev_out[2 * tempId + 1 + bitIndex];
//        cuDecaFFTCoalesce[tempId] = v3;



        int i = 0;
        offset = i * Ns2;
        aID = keySI + offset + id % Ns2;
        bID = keySI + offset + id % Ns2 + Ns2 * kpl;
        hipfftDoubleComplex temp_a0 = hipCmul(v0, bki[aID]);
        hipfftDoubleComplex temp_b0 = hipCmul(v0, bki[bID]);

        i = 1;
        offset = i * Ns2;
        aID = keySI + offset + id % Ns2;
        bID = keySI + offset + id % Ns2 + Ns2 * kpl;
        hipfftDoubleComplex temp_a1 = hipCmul(v1, bki[aID]);
        hipfftDoubleComplex temp_b1 = hipCmul(v1, bki[bID]);

        i = 2;
        offset = i * Ns2;
        aID = keySI + offset + id % Ns2;
        bID = keySI + offset + id % Ns2 + Ns2 * kpl;
        hipfftDoubleComplex temp_a2 = hipCmul(v2, bki[aID]);
        hipfftDoubleComplex temp_b2 = hipCmul(v2, bki[bID]);

        i = 3;
        offset = i * Ns2;
        aID = keySI + offset + id % Ns2;
        bID = keySI + offset + id % Ns2 + Ns2 * kpl;
        hipfftDoubleComplex temp_a3 = hipCmul(v3, bki[aID]);
        hipfftDoubleComplex temp_b3 = hipCmul(v3, bki[bID]);

        hipfftDoubleComplex tmpa_gpuCoal0;
        tmpa_gpuCoal0.x = temp_a0.x + temp_a1.x + temp_a2.x + temp_a3.x;
        tmpa_gpuCoal0.y = temp_a0.y + temp_a1.y + temp_a2.y + temp_a3.y;
//        tmpa_gpuCoal[id] = tmpa_gpuCoal0;

        hipfftDoubleComplex tmpa_gpuCoal1;
        tmpa_gpuCoal1.x = temp_b0.x + temp_b1.x + temp_b2.x + temp_b3.x;
        tmpa_gpuCoal1.y = temp_b0.y + temp_b1.y + temp_b2.y + temp_b3.y;
//        tmpa_gpuCoal[id + Ns2] = tmpa_gpuCoal1;

        register int largeSI = (id / Ns2) * (N + 1);
        register int tid = id % Ns2;
        d_in[largeSI + 2 * tid + 1] = tmpa_gpuCoal0;

        largeSI = (id / Ns2 + 1) * (N + 1);
        d_in[largeSI + 2 * tid + 1] = tmpa_gpuCoal1;




        //init with 0
//        tmpa_gpuCoal[id].x = 0;
//        tmpa_gpuCoal[id].y = 0;
//        tmpa_gpuCoal[Ns2 + id].x = 0;
//        tmpa_gpuCoal[Ns2 + id].y = 0;
//#pragma unroll
//        for (int i = 0; i < kpl; ++i) {//kpl
//            offset = i * Ns2;
//            aID = keySI + offset + id;
//            bID = keySI + offset + id + Ns2 * kpl;
//
//            hipfftDoubleComplex temp_a = hipCmul(cuDecaFFTCoalesce[offset + id], bki[aID]);
//            tmpa_gpuCoal[id].x += temp_a.x;
//            tmpa_gpuCoal[id].y += temp_a.y;
//
//            hipfftDoubleComplex temp_b = hipCmul(cuDecaFFTCoalesce[offset + id], bki[bID]);
//            tmpa_gpuCoal[Ns2 + id].x += temp_b.x;
//            tmpa_gpuCoal[Ns2 + id].y += temp_b.y;
//
//        }
//    }
//    __syncthreads();
//    __syncthreads();
//    __syncthreads();
//    __syncthreads();
//    __syncthreads();
//    __syncthreads();

//    if (id < 1024) {
//        register int largeSI = (id / Ns2) * (N + 1);
//        register int tid = id % Ns2;
//        d_in[largeSI + 2 * tid + 1] = tmpa_gpuCoal[id];
////        d_in[largeSI + 2 * tid + 1].y = 1;//tmpa_gpuCoal[id];
//    }

}


__global__ void finishUpFFT_n_Bit(int *temp2, hipfftDoubleReal *d_out, int *temp3) {
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    register int N = 1024, _2N = 2048;
    register double _2p32 = double(INT64_C(1) << 32);
    register double _1sN = double(1) / double(N);
    register int bitIndex = id / N;
    register int tIndex = id % N;
    register int startIndexLarge = bitIndex * _2N;
    temp2[id] = Torus32(int64_t(d_out[startIndexLarge + tIndex] * _1sN * _2p32)) + temp3[id];//

}




__global__ void extractionAndKeySwitch_1_Bit(int *result_a, int *result_b,
                                       uint32_t *coal_d_aibar, uint32_t  *coal_d_aij,
                                       int *accum_a_b,
                                       Torus32 *ks_a_gpu_extendedPtr, Torus32 *ks_b_gpu_extendedPtr,
                                       double *ks_cv_gpu_extendedPtr) {
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    register int N = 1024, _2N = 2048, basebit = 2, base = 1 << basebit, mask = base - 1, t =8;
    register int32_t prec_offset = 1 << (32 - (1 + basebit * t));


    register int index = 0;
    register int bitIndex = id / N;
    register int tIndex = id % N;//corresponding to j
    register int startIndex = bitIndex * N;

    __shared__ uint32_t s_coal_d_aibar[1024];
//    __shared__ uint32_t coal_d_aij[1024 * 8];


    bool multipleOfN = id % N == 0;
    s_coal_d_aibar[id] = (multipleOfN) * (accum_a_b[index - tIndex + startIndex] + prec_offset)
            + (!multipleOfN) * (-accum_a_b[index - tIndex + startIndex + N] + prec_offset);


//    if (id % N == 0) {
//        coal_d_aibar[id] = accum_a_b[index - tIndex + startIndex] + prec_offset;
//    } else {
//        coal_d_aibar[id] = -accum_a_b[index - tIndex + startIndex + N] + prec_offset;
//    }


    __syncthreads();
//    __syncthreads();
//    __syncthreads();
//    __syncthreads();
//    __syncthreads();
//    __syncthreads();
//    __syncthreads();
//    __syncthreads();
//    if(id < 1024) {//t
        register int tempID = id;
        register int i = tempID / t;
        register int j = tempID % t;
        coal_d_aij[tempID] = (s_coal_d_aibar[i] >> (32 - (j + 1) * basebit)) & mask;

        tempID += 1024;
        i = tempID / t;
        j = tempID % t;
        coal_d_aij[tempID] = (s_coal_d_aibar[i] >> (32 - (j + 1) * basebit)) & mask;

        tempID += 1024;
        i = tempID / t;
        j = tempID % t;
        coal_d_aij[tempID] = (s_coal_d_aibar[i] >> (32 - (j + 1) * basebit)) & mask;

        tempID += 1024;
        i = tempID / t;
        j = tempID % t;
        coal_d_aij[tempID] = (s_coal_d_aibar[i] >> (32 - (j + 1) * basebit)) & mask;

        tempID += 1024;
        i = tempID / t;
        j = tempID % t;
        coal_d_aij[tempID] = (s_coal_d_aibar[i] >> (32 - (j + 1) * basebit)) & mask;

        tempID += 1024;
        i = tempID / t;
        j = tempID % t;
        coal_d_aij[tempID] = (s_coal_d_aibar[i] >> (32 - (j + 1) * basebit)) & mask;

        tempID += 1024;
        i = tempID / t;
        j = tempID % t;
        coal_d_aij[tempID] = (s_coal_d_aibar[i] >> (32 - (j + 1) * basebit)) & mask;

        tempID += 1024;
        i = tempID / t;
        j = tempID % t;
        coal_d_aij[tempID] = (s_coal_d_aibar[i] >> (32 - (j + 1) * basebit)) & mask;
//    }

//    __syncthreads();
    int subFromB = 0;
    int bi;
    if (id < 500) {
        result_a[id] = 0;
        register int A = 1024, B = t, C = base, D = 500, ks_t = 8;
#pragma unroll 0
        for (int i = 0; i < 1024; ++i) {
            int sI =  i * ks_t;
#pragma unroll 0
            for (int j = 0; j < 8; ++j) {//ks_t
                int sI2 = sI + j;
                int aij = coal_d_aij[sI2];
                if (aij != 0) {
                    result_a[id] -= ks_a_gpu_extendedPtr[i * B * C * D + j * C * D + aij * D + (id % D)];//sourceA[(i * B * C * D + j * C * D+ aij * params_n +  id)];//source[aij][id];
                }
//                if(id < 1) {
                    bi = coal_d_aij[sI2 + id];
                    subFromB += ks_b_gpu_extendedPtr[i * B * C + j * C + bi];
//                }
            }
        }
    }

    if (id < 1) {
        result_b[0] = accum_a_b[N] - subFromB;
    }


}


void bootstrapping_gull_gpu_1_bit_wise(LweSample_16 *result, int *temp_res_a, int *temp_res_b, int nBits,
                                       hipfftDoubleComplex *cudaBkFFTCoalesceExt,
                                       Torus32 *ks_a_gpu_extendedPtr,
                                       Torus32 *ks_b_gpu_extendedPtr, double *ks_cv_gpu_extendedPtr) {

    //bootstrapping woks uptoFFT
    int nThreads = 1024, BLOCKSIZE = 1024, k = 1, N = 1024, kpl = 4, Ns2 = 512, _2N = 2048;
    static const Torus32 MU = modSwitchToTorus32(1, 8);

    int gridSize = (int) ceil((float) (nThreads) / BLOCKSIZE);//1

    int *accum_a_b, *bara, *temp_accum_a_b;//accum a and accum b together; bara; tempaccum for mux rotate
    hipMalloc(&accum_a_b, nBits * 1024 * (k + 1) * sizeof(int));
    hipMalloc(&temp_accum_a_b, nBits * 1024 * (k + 1) * sizeof(int));
    hipMalloc(&bara, nBits * 1024 * sizeof(int));


    hipDeviceProp_t cProfile;
    hipGetDeviceProperties(&cProfile, 0);
    int nSM = cProfile.multiProcessorCount;
    cout << "#SM: " << nSM << endl; //20
    hipStream_t streams[nSM];

#pragma unroll
    for (int i = 0; i < 20; ++i) {//nSM
        hipStreamCreateWithFlags(&streams[i], hipStreamNonBlocking);
    }

    for (int bIndex = 0; bIndex < nBits; ++bIndex) {
        int accumStart = bIndex * (1024 * (k + 1));
        int baraStart = bIndex * 1024;
        int temp_res_aStart = bIndex * 500;

        bootstrappingUptoBlindRotate_OneBit<<<gridSize, BLOCKSIZE, 0, streams[bIndex % nSM]>>>
                                                                      (accum_a_b + accumStart,
                                                                              temp_accum_a_b + accumStart,
                                                                              bara + baraStart,
                                                                              MU,
                                                                              temp_res_a + temp_res_aStart,
                                                                              temp_res_b[bIndex], NULL,
                                                                              cudaBkFFTCoalesceExt);
    }
    hipDeviceSynchronize();


    //after blind rotate
    int *decaCoalesce;
    hipMalloc(&decaCoalesce, nBits * N * kpl * sizeof(int));//1024*4

    hipfftDoubleComplex *cuDecaFFTCoalesce;
    hipMalloc(&cuDecaFFTCoalesce, nBits * kpl * Ns2 * sizeof(hipfftDoubleComplex));//512*4

    hipfftDoubleComplex *tmpa_gpuCoal;
    hipMalloc(&tmpa_gpuCoal, nBits * Ns2 * sizeof(hipfftDoubleComplex) * (k + 1));

    //fft variables
    hipfftDoubleReal* d_rev_in;
    hipfftDoubleComplex *d_rev_out;
    hipfftDoubleComplex *d_in;
    hipfftDoubleReal *d_out;
    int batch = kpl;
    int dParts = 4;
    //fft plans
    hipfftHandle p;
    hipfftHandle rev_p;
    //fft variables allocation
    hipMalloc(&d_rev_in, nBits * sizeof(hipfftDoubleReal) * _2N * batch);
    hipMalloc(&d_rev_out, nBits * sizeof(hipfftDoubleComplex) * (N + 1) * batch);
    hipfftPlan1d(&rev_p, _2N, HIPFFT_D2Z, nBits * batch);//(nBits * batch)/dParts);// (batch - (batch/dParts)));

    batch = 2;//batch/dParts;//a and b together
    hipMalloc(&d_in, nBits * sizeof(hipfftDoubleComplex) * (N + 1) * batch);//batch
    hipMalloc(&d_out, nBits * sizeof(hipfftDoubleReal) * _2N * batch);
    hipfftPlan1d(&p, _2N, HIPFFT_Z2D, nBits * batch);

    int *temp2 = temp_accum_a_b;
    int *temp3 = accum_a_b;


//    assert(nBits == 1);

    //call tfhe_MuxRotate_FFT_16
#pragma unroll
    for (int j = 0; j < 500; ++j) {//500
        gridSize = 2;//2;//as accum is of (k + 1)

        for (int bIndex = 0; bIndex < nBits; ++bIndex) {
            //find starting indices
            int accumStart = bIndex * 1024 * (k + 1);
            int decaCoalesceStart = bIndex * 1024 * kpl;
            int d_rev_inStart = bIndex * _2N * kpl;
            int baraStart = bIndex * N;
            prepareForiFFT_1_Bit<<<gridSize, BLOCKSIZE, 0, streams[bIndex % nSM]>>>
                                                           (temp2 + accumStart,
                                                                   decaCoalesce + decaCoalesceStart,
                                                                   d_rev_in + d_rev_inStart,
                                                                   bara + baraStart,
                                                                   j,
                                                                   temp3 + accumStart);
        }
        hipDeviceSynchronize();

        hipfftExecD2Z(rev_p, d_rev_in, d_rev_out);
        hipDeviceSynchronize();

        int length = kpl * Ns2;//4 * 512 = 2048
        gridSize = 1;//(int) ceil((float) (length) / BLOCKSIZE); //2
        for (int bIndex = 0; bIndex < nBits; ++bIndex) {
            int cuDecaFFTCoalesceStart = bIndex * kpl * Ns2;
            int tmpa_gpuCoalStart = bIndex * (k + 1) * Ns2;
            int d_inStart = bIndex * (N + 1) * (k + 1);
            int d_rev_outStart = bIndex *(N + 1) * kpl;
            prepareForFFT_1_Bit<<<gridSize, 512, 0, streams[bIndex % nSM]>>>
                                                    (cuDecaFFTCoalesce + cuDecaFFTCoalesceStart,
                                                            tmpa_gpuCoal + tmpa_gpuCoalStart,
                                                            d_in + d_inStart,
                                                            d_rev_out + d_rev_outStart,
                                                            cudaBkFFTCoalesceExt, j, N, Ns2, length);
        }
        hipDeviceSynchronize();



        hipfftExecZ2D(p, d_in, d_out);
        hipDeviceSynchronize();


        //after fft
        length = N * 2;
        gridSize = (int) ceil((float) (length) / BLOCKSIZE); //2
        for (int bIndex = 0; bIndex < nBits; ++bIndex) {
            int accumStart = bIndex * 1024 * (k + 1);
            int d_outStart = bIndex * _2N * (k + 1);
            finishUpFFT_n_Bit<<<gridSize, BLOCKSIZE, 0, streams[bIndex % nSM]>>>
                                                        (temp2 + accumStart,
                                                                d_out + d_outStart,
                                                                temp3 + accumStart);
        }
        hipDeviceSynchronize();

        swap(temp2, temp3);
    }




    //output is in temp3
    //extract and ks
    //intermediate variables to test u (delete afterwards)
    int *result_b;
    double *result_cv = NULL;
    hipMalloc(&result_b, nBits * sizeof(int));
//    hipMalloc(&result_cv, 1 * sizeof(double));


    uint32_t *coal_d_aibar;
    hipMalloc(&coal_d_aibar, nBits * N * sizeof(uint32_t));

    int coal_d_aijSize = nBits * N * 8;//t
    uint32_t  *coal_d_aij;
    hipMalloc(&coal_d_aij, coal_d_aijSize * sizeof(uint32_t));

//    int length = N * 8;//t
    gridSize = 1;//(int) ceil((float) (length) / BLOCKSIZE);
    for (int bIndex = 0; bIndex < nBits; ++bIndex) {

        int result_aStart = bIndex * 500;
        int result_bStart = bIndex;
        int coal_d_aibarStart = bIndex * N;
        int coal_d_aijStart = bIndex * N * 8;
        int accumStart = bIndex * (k + 1) * 1024;


        extractionAndKeySwitch_1_Bit<<<gridSize, BLOCKSIZE, 0, streams[bIndex % nSM]>>>
                                                               (result->a + result_aStart,
                                                                       result_b + result_bStart,
                                                                       coal_d_aibar + coal_d_aibarStart,
                                                                       coal_d_aij + coal_d_aijStart,
                                                                       temp3 + accumStart,
                                                                       ks_a_gpu_extendedPtr,
                                                                       ks_b_gpu_extendedPtr,
                                                                       ks_cv_gpu_extendedPtr);
    }
    hipDeviceSynchronize();

    hipMemcpy(result->b, result_b, nBits * sizeof(int), hipMemcpyDeviceToHost);


//    int *temp = new int[500];
//    hipMemcpy(temp, result->a, 500 * sizeof(int), hipMemcpyDeviceToHost);
//    for (int i = 0; i < 500; ++i) {
//        cout << temp[i] << " ";
//    }
//    cout << endl;
//    cout << result->b[0] << endl;

//    assert(nBits == 1);

#pragma unroll
    for (int i = 0; i < 20; ++i) { //nSM
        hipStreamDestroy(streams[i]);

    }

    hipFree(temp_res_a);
    hipFree(accum_a_b);
    hipFree(temp_accum_a_b);
    hipFree(bara);
    hipFree(decaCoalesce);//1024*4
    hipFree(cuDecaFFTCoalesce);//512*4
    hipFree(tmpa_gpuCoal);
    hipFree(d_rev_in);
    hipFree(d_rev_out);
    hipFree(d_in);//batch
    hipFree(d_out);
    hipFree(result_b);
    hipFree(coal_d_aibar);
    hipFree(coal_d_aij);

    hipfftDestroy(rev_p);
    hipfftDestroy(p);
}






EXPORT void bootsAND_fullGPU_OneBit(LweSample_16 *result, const LweSample_16 *ca, const LweSample_16 *cb, int nBits,
                                    hipfftDoubleComplex *cudaBkFFTCoalesceExt, Torus32 *ks_a_gpu_extendedPtr,
                                    Torus32 *ks_b_gpu_extendedPtr, double *ks_cv_gpu_extendedPtr) {

    const int n = 500, BLOCKSIZE = 1024, N = 1024, _2N = 2048, Ns2 = 512, k = 1, kpl = 4, l = 2, offset = 2149580800,
            halfBg = 512, maskMod = 1023;

    static const Torus32 MU = modSwitchToTorus32(1, 8);
    //compute: (0,-1/8) + ca + cb
    static const Torus32 AndConst = modSwitchToTorus32(-1, 8);

    int *temp_res_a, *temp_res_b;
    hipMalloc(&temp_res_a, n * nBits * sizeof(Torus32));
    temp_res_b = new int[nBits];

    int gridSize = (int) ceil((float) (n * nBits) / BLOCKSIZE);

    vecAdd<<<gridSize, BLOCKSIZE>>>(temp_res_a, ca->a, cb->a, n * nBits);
    for (int i = 0; i < nBits; ++i) {
        temp_res_b[i] = ca->b[i] + cb->b[i] + AndConst;
    }

    bootstrapping_gull_gpu_1_bit_wise(result, temp_res_a, temp_res_b, nBits, cudaBkFFTCoalesceExt,
                                      ks_a_gpu_extendedPtr, ks_b_gpu_extendedPtr, ks_cv_gpu_extendedPtr);

    hipFree(temp_res_a);
    delete [] temp_res_b;
}





























__global__ void bootstrappingUptoBlindRotate_n_Bit(int *accum_a_b, int *temp_accum_a_b, int *bara, int *testvectbis,
                                                   Torus32 MU, int nBits,
                                                   int *temp_res_a, int *barb) {
    register int id = blockIdx.x * blockDim.x + threadIdx.x;
    register int n = 500, N = 1024, _2N = 2048, Ns2 = 512, Nx2 = 2048;

    register int bIndex = id / N;
    register int baraIndex = id % N;
    register int barbi = barb[bIndex];
    //torusPolynomialMulByXai_16 -> res ->testvectbis, v-> torusPolyTestvect_coef
    register int a = _2N - barbi;
    register int aa = a - N;
    testvectbis[id] = MU;

    if (a < N) {//1024
        if (baraIndex < a) {
            testvectbis[id] = -MU;
        }
    } else {
        if (baraIndex >= aa) {
            testvectbis[id] = -MU;
        }
    }
    for (int i = 0; i < nBits; ++i) {
        __syncthreads();
    }
    accum_a_b[id] = 0;//accum_a
    accum_a_b[1024 * nBits + id] = testvectbis[id];

    temp_accum_a_b[id] = 0;//accum_a
    temp_accum_a_b[1024 * nBits + id] = 0;

    bara[id] = 0;

    if (id < nBits * 500) {//500
        bIndex = id / 500;
        register  int destinationIndex = bIndex * N + id % n;
        bara[destinationIndex] = modSwitchFromTorus32_GPU_device(temp_res_a[id], Nx2);
    }
}



__global__ void prepareForiFFT_n_Bit(int *des, int *decaCoalesce, hipfftDoubleReal *d_rev_in,
                                     int nBits, int nGrid, int *bara, int baraIndex, int *source) {
    register int id = blockIdx.x * blockDim.x + threadIdx.x;
    register int N = 1024, _2N = 2048, Ns2 = 512;

//    bool outerBlock = id < nBits * 2 * 1024;
//    if (id < nBits * 2 * 1024) {//nBits * (k + 1) * 1024
        register int bitIndex = (id / N) % nBits;
        register int threadIdModN = id % N;

        register int a = bara[bitIndex * N + baraIndex];
        register int aa = a - N;

        register bool l1 = a < N, l2 = threadIdModN < a, l3 = threadIdModN < aa;

//    des[id] = (!outerBlock) * des[id]
//              + outerBlock * (l1 * (l2 * (-source[id - a + N] - source[id])
//                                    + (!l2) * (source[id - a] - source[id]))
//                              + (!l1) * (l3 * (source[id - aa + N] - source[id])
//                                         + (!l3) * (-source[id - aa] - source[id])));

    int des_id = (l1 * (l2 * (-source[id - a + N] - source[id])
                                       + (!l2) * (source[id - a] - source[id]))
                                 + (!l1) * (l3 * (source[id - aa + N] - source[id])
                                            + (!l3) * (-source[id - aa] - source[id])));

//        if (a < N) {
//            if (threadIdModN < a) {
//                des[id] = -source[id - a + N] - source[id];
//            } else {
//                des[id] = source[id - a] - source[id];
//            }
//        } else {
//            if (threadIdModN < aa) {
//                des[id] = source[id - aa + N] - source[id];
//            } else {
//                des[id] = -source[id - aa] - source[id];
//            }
//        }
//    }
//    for (int i = 0; i < nGrid; ++i) {
//        __syncthreads();
//    }

    register uint32_t halfBg = 512, maskMod = 1023, Bgbit = 10, kpl = 4, l = 2;
    register uint32_t offset = 2149580800;

    bool middleBlock = id < nBits * 2 * 1024;//4//kpl

//    if(id < nBits * 4 * 1024) {//nBits * kpl * 1024
//        register int p = (id/(N * nBits)) % l;//0 1 0 1
//        register int index = (id/(l * N * nBits)) % l;//0 1

//    register int sI = index * N * nBits;
//    register int tid2 = id % (N * nBits);

//        register uint32_t val = ((uint32_t)(des[sI + tid2] + offset));



//        decaCoalesce[id] = middleBlock * (id) + (!middleBlock) * (decaCoalesce[id]);// middleBlock * (temp1 - halfBg) + (!middleBlock) * decaCoalesce[id];
    register int p = 0;
    register int decal = (32 - (p + 1) * Bgbit);
    register uint32_t val = ((uint32_t)(des_id + offset));
    register uint32_t temp1 = (val >> decal) & maskMod;

    register int xxxxx1 = (temp1 - halfBg);// + (!middleBlock) * (decaCoalesce[((id / (N * nBits)) * (N * nBits)) + id]);

    decaCoalesce[((id / (N * nBits)) * (N * nBits)) + id] = xxxxx1;
//            middleBlock * (temp1 - halfBg) +
//            (!middleBlock) * (decaCoalesce[((id / (N * nBits)) * (N * nBits)) + id]);


    p = 1;
    decal = (32 - (p + 1) * Bgbit);
    val = ((uint32_t)(des_id + offset));
    temp1 = (val >> decal) & maskMod;

    register int xxxxx2 = temp1 - halfBg;// +

    decaCoalesce[((id / (N * nBits)) * (N * nBits)) + id + (N * nBits)] = xxxxx2;
//            middleBlock * (temp1 - halfBg) +
//            (!middleBlock) * decaCoalesce[((id / (N * nBits)) * (N * nBits)) + id + (N *
//                                                                                     nBits)];//(temp1 - halfBg) + (!middleBlock) * decaCoalesce[id];//middleBlock * id;//middleBlock * id; //1;//middleBlock * (id) + (!middleBlock) * (decaCoalesce[id]);// middleBlock * (temp1 - halfBg) + (!middleBlock) * decaCoalesce[id];
                          //(!middleBlock) * decaCoalesce[((id / (N * nBits)) * (N * nBits)) + id + (N * nBits)];
//        decaCoalesce[(nBits * N) + id] = middleBlock * id;//middleBlock * id; //1;//middleBlock * (id) + (!middleBlock) * (decaCoalesce[id]);// middleBlock * (temp1 - halfBg) + (!middleBlock) * decaCoalesce[id];
//        decaCoalesce[(nBits * N) + id] = middleBlock * id;//middleBlock * id; //1;//middleBlock * (id) + (!middleBlock) * (decaCoalesce[id]);// middleBlock * (temp1 - halfBg) + (!middleBlock) * decaCoalesce[id];
//    }

//    for (int i = 0; i < nGrid; ++i) {
//        __syncthreads();
//    }
    register int bIndex = id / _2N, tIndex = id % _2N, startIndexSmall = bIndex * N;

//    middleBlock = tIndex < N;

//    d_rev_in[id] = middleBlock * (decaCoalesce[startIndexSmall + tIndex] / 2.)
//            + (!middleBlock) * (d_rev_in[id] = -decaCoalesce[startIndexSmall + tIndex - N] / 2.);


//    d_rev_in[((id / (N * nBits)) * (N * nBits)) + id + (tIndex >= N) * 1024 * bIndex] = id;//middleBlock * (1) + (!middleBlock) * (d_rev_in[id]);
        bIndex = id / N;
        tIndex = id % N;
        int destTod_rev_in = bIndex * _2N + tIndex + (bIndex >= nBits) * nBits * N * 2;
        d_rev_in[destTod_rev_in] = xxxxx1/2.;//id;//
        d_rev_in[destTod_rev_in + 1024] = -xxxxx1/2.;//id;//

        destTod_rev_in += nBits * 2 * 1024;
        d_rev_in[destTod_rev_in] = xxxxx2/2.;//id;
        d_rev_in[destTod_rev_in + 1024] = -xxxxx2/2.;//id;
//        d_rev_in[bIndex * _2N + tIndex + nBits * 4 * 1024] = 3;//xxxxx2/2.;
//        d_rev_in[bIndex * _2N + tIndex + 1024 + nBits * 4 * 1024] = 4;//-xxxxx2/2.;


//        d_rev_in[id + (N * nBits * 2) * 2] = id;

//    d_rev_in[((id / (N * nBits)) * (N * nBits)) + id + (tIndex < N) * 1024] = id;//middleBlock * (1) + (!middleBlock) * (d_rev_in[id]);
//    d_rev_in[((id / (N * nBits)) * (N * nBits)) + id + (tIndex < N) * 1024] = id;//middleBlock * (1) + (!middleBlock) * (d_rev_in[id]);
//    d_rev_in[((id / (N * nBits)) * (N * nBits)) + id + (N * 2 * nBits) + (tIndex >= N) * 1024] = id;
//    d_rev_in[((id / (N * nBits)) * (N * nBits)) + id + (N * 2 * nBits * 2) + (tIndex >= N) * 1024] = id; //middleBlock * (1) + (!middleBlock) * (d_rev_in[id]);
//    d_rev_in[((id / (N * nBits)) * (N * nBits)) + id + (N * 2 * nBits * 3) + (tIndex >= N) * 1024] = id + (N * 2 * nBits); //middleBlock * (1) + (!middleBlock) * (d_rev_in[id]);
//    d_rev_in[((id / (N * nBits)) * (N * nBits)) + id + (N * 2 * nBits * 3)] = id; //middleBlock * (1) + (!middleBlock) * (d_rev_in[id]);
//    d_rev_in[((id / (N * nBits)) * (N * nBits)) + id + (N * nBits)] = id; //middleBlock * (1) + (!middleBlock) * (d_rev_in[id]);
//    d_rev_in[nBits * N + id] = middleBlock * (2) + (!middleBlock) * (d_rev_in[id]);

//    if (tIndex < N) {
//        d_rev_in[id] = decaCoalesce[startIndexSmall + tIndex] / 2.;
//    } else {
//        d_rev_in[id] = -decaCoalesce[startIndexSmall + tIndex - N] / 2.;
//    }

}




__global__  void prepareForFFT_n_Bit(hipfftDoubleComplex *cuDecaFFTCoalesce, hipfftDoubleComplex *tmpa_gpuCoal,
                                     hipfftDoubleComplex *d_in, hipfftDoubleComplex *d_rev_out,
                                     hipfftDoubleComplex *bki,  int keyIndex, int nBits, int nGrid) {
    register int id = blockIdx.x*blockDim.x+threadIdx.x;
    register int Ns2 = 512;
//    if (id < nBits * 4 * 512) {//nBits * kpl * Ns2
    int tempId = id;
    int bitIndex = tempId/Ns2;
    register hipfftDoubleComplex v0 = d_rev_out[2 * tempId + 1 + bitIndex];//d_rev_out[2 * id + 1 + bitIndex];
//    cuDecaFFTCoalesce[tempId] = v0;

    tempId = tempId + (Ns2 * nBits);
    bitIndex = (tempId)/Ns2;
    register hipfftDoubleComplex v1 = d_rev_out[2 * tempId + 1 + bitIndex];
//    cuDecaFFTCoalesce[tempId] = v1;

    tempId = tempId + (Ns2 * nBits);
    bitIndex = (tempId)/Ns2;
    register hipfftDoubleComplex v2 = d_rev_out[2 * tempId + 1 + bitIndex];
//    cuDecaFFTCoalesce[tempId] = v2;

    tempId = tempId + (Ns2 * nBits);
    bitIndex = (tempId)/Ns2;
    register hipfftDoubleComplex v3 = d_rev_out[2 * tempId + 1 + bitIndex];
//    cuDecaFFTCoalesce[tempId] = v3;

//    }
    register int k = 1, kpl = 4, keySI = keyIndex * (k + 1) * kpl * Ns2, aID, bID, offset;
//
//    if (id < nBits * 512) {
//        tmpa_gpuCoal[id].x = 0;
//        tmpa_gpuCoal[id].y = 0;
//        tmpa_gpuCoal[nBits * Ns2 + id].x = 0;
//        tmpa_gpuCoal[nBits * Ns2 + id].y = 0;


    int i = 0;
    offset = i * Ns2;
    aID = keySI + offset + id % Ns2;
    bID = keySI + offset + id % Ns2 + Ns2 * kpl;
    hipfftDoubleComplex temp_a0 = hipCmul(v0, bki[aID]);
    hipfftDoubleComplex temp_b0 = hipCmul(v0, bki[bID]);

    i = 1;
    offset = i * Ns2;
    aID = keySI + offset + id % Ns2;
    bID = keySI + offset + id % Ns2 + Ns2 * kpl;
    hipfftDoubleComplex temp_a1 = hipCmul(v1, bki[aID]);
    hipfftDoubleComplex temp_b1 = hipCmul(v1, bki[bID]);

    i = 2;
    offset = i * Ns2;
    aID = keySI + offset + id % Ns2;
    bID = keySI + offset + id % Ns2 + Ns2 * kpl;
    hipfftDoubleComplex temp_a2 = hipCmul(v2, bki[aID]);
    hipfftDoubleComplex temp_b2 = hipCmul(v2, bki[bID]);


    i = 3;
    offset = i * Ns2;
    aID = keySI + offset + id % Ns2;
    bID = keySI + offset + id % Ns2 + Ns2 * kpl;
    hipfftDoubleComplex temp_a3 = hipCmul(v3, bki[aID]);
    hipfftDoubleComplex temp_b3 = hipCmul(v3, bki[bID]);




//    tmpa_gpuCoal[id] = temp_a3;
//    tmpa_gpuCoal[nBits * Ns2 + id] = temp_b3;



    hipfftDoubleComplex tmpa_gpuCoal0;
    tmpa_gpuCoal0.x = temp_a0.x + temp_a1.x +temp_a2.x +temp_a3.x;
    tmpa_gpuCoal0.y = temp_a0.y + temp_a1.y +temp_a2.y +temp_a3.y;
//    tmpa_gpuCoal[id] = tmpa_gpuCoal0;

    hipfftDoubleComplex tmpa_gpuCoal1;
    tmpa_gpuCoal1.x = temp_b0.x + temp_b1.x +temp_b2.x +temp_b3.x;
    tmpa_gpuCoal1.y = temp_b0.y + temp_b1.y +temp_b2.y +temp_b3.y;
//    tmpa_gpuCoal[nBits * Ns2 + id] = tmpa_gpuCoal1;


//    hipfftDoubleComplex temp_a = hipCmul(cuDecaFFTCoalesce[i * (Ns2 * nBits) + id], bki[aID]);
//    hipfftDoubleComplex temp_b = hipCmul(cuDecaFFTCoalesce[i * (Ns2 * nBits) + id], bki[bID]);

//
//    for (int i = 0; i < nGrid * 4; ++i) {
//        __syncthreads();
//    }
//    if (id < nBits * 2 * Ns2) {//nBits * (k + 1) * Ns2
        register int N = 1024, largeSI = (id / Ns2) * (N + 1);
        register int tid = id % Ns2;
        d_in[largeSI + 2 * tid + 1] = tmpa_gpuCoal0;

    largeSI = (id / Ns2 + nBits) * (N + 1);
        d_in[largeSI + 2 * tid + 1] = tmpa_gpuCoal1;

//        d_in[largeSI + 2 * tid + 1].y = tmpa_gpuCoal[id].y;
//    }
        __syncthreads();

}

__global__ void finishUpFFT_n_Bit(int *temp2, hipfftDoubleReal *d_out, int *temp3, int nBits) {

    register int id = blockIdx.x*blockDim.x+threadIdx.x;
    register int N = 1024, _2N = 2048;
    register double _2p32 = double(INT64_C(1) << 32);
    register double _1sN = double(1) / double(N);
    register int bitIndex = id / N;
    register int tIndex = id % N;
    register int startIndexLarge = bitIndex * _2N;
    temp2[id] = Torus32(int64_t(d_out[startIndexLarge + tIndex] * _1sN * _2p32)) + temp3[id];
//    if(id < nBits * 2048) {
//    }
        __syncthreads();
}


EXPORT void bootsAND_fullGPU_n_Bit(LweSample_16 *result, const LweSample_16 *ca, const LweSample_16 *cb, int nBits,
                                    hipfftDoubleComplex *cudaBkFFTCoalesceExt, Torus32 *ks_a_gpu_extendedPtr,
                                    Torus32 *ks_b_gpu_extendedPtr, double *ks_cv_gpu_extendedPtr) {

    register int n = 500, BLOCKSIZE = 1024, N = 1024, _2N = 2048, Ns2 = 512, k = 1, kpl = 4, l = 2, offset = 2149580800,
            halfBg = 512, maskMod = 1023;
    cout << "bBits: " << endl;
    cout << "here" << endl;

    static const Torus32 MU = modSwitchToTorus32(1, 8);
    //compute: (0,-1/8) + ca + cb
    static const Torus32 AndConst = modSwitchToTorus32(-1, 8);

    int *temp_res_a, *temp_res_b;
    hipMalloc(&temp_res_a, n * nBits * sizeof(Torus32));
    temp_res_b = new Torus32[nBits];


    register int length = 500 * nBits;
    int gridSize = (int) ceil((float) (length) / BLOCKSIZE);

    vecAdd<<<gridSize, BLOCKSIZE>>>(temp_res_a, ca->a, cb->a, length);
    for (int i = 0; i < nBits; ++i) {
        temp_res_b[i] = ca->b[i] + cb->b[i] + AndConst;
        temp_res_b[i] = modSwitchFromTorus32(temp_res_b[i], _2N);
    }
//    temp_res_cv += ca->current_variance[0] + cb->current_variance[0];

    //bootstrapping woks uptoFFT
    int *accum_a_b, *bara, *temp_accum_a_b, *barb, *testvectbis;//accum a and accum b together; bara; tempaccum for mux rotate
    hipMalloc(&accum_a_b, nBits * N * (k + 1) * sizeof(int));
    hipMalloc(&temp_accum_a_b, nBits * N * (k + 1) * sizeof(int));
    hipMalloc(&bara, nBits * N * sizeof(int));
    hipMalloc(&barb, nBits * sizeof(int));
    hipMalloc(&testvectbis, nBits * N * sizeof(int));
    hipMemcpy(barb, temp_res_b, nBits * sizeof(int), hipMemcpyHostToDevice);

    gridSize = nBits;
    bootstrappingUptoBlindRotate_n_Bit<<<gridSize, BLOCKSIZE>>>(accum_a_b, temp_accum_a_b, bara, testvectbis, MU, nBits, temp_res_a, barb);

    int *decaCoalesce;
    hipMalloc(&decaCoalesce, nBits * N * kpl * sizeof(int));//1024*4

    hipfftDoubleComplex *cuDecaFFTCoalesce;
    hipMalloc(&cuDecaFFTCoalesce, nBits * kpl * Ns2 * sizeof(hipfftDoubleComplex));//512*4

    hipfftDoubleComplex *tmpa_gpuCoal;
    hipMalloc(&tmpa_gpuCoal, nBits * Ns2 * sizeof(hipfftDoubleComplex) * (k + 1));//512*2

    //fft variables
    int iFFTBatch = nBits * kpl;
    int FFTBatch = nBits * (k + 1);
    int dParts = 4;
    //cufft helper variables
    hipfftDoubleReal* d_rev_in;
    hipfftDoubleComplex *d_rev_out;
    hipfftDoubleComplex *d_in;
    hipfftDoubleReal *d_out;
    //cufft plans
    hipfftHandle p;
    hipfftHandle rev_p;
    //ifft variables allocation
    hipMalloc(&d_rev_in, iFFTBatch * _2N * sizeof(hipfftDoubleReal));
    hipMalloc(&d_rev_out, iFFTBatch * (N + 1) * sizeof(hipfftDoubleComplex));
    hipfftPlan1d(&rev_p, _2N, HIPFFT_D2Z, iFFTBatch);// - (iFFTBatch / dParts));
    //fft variables allocation
    hipMalloc(&d_in, FFTBatch * (N + 1) * sizeof(hipfftDoubleComplex));
    hipMalloc(&d_out, FFTBatch * _2N * sizeof(hipfftDoubleReal));
    hipfftPlan1d(&p, _2N, HIPFFT_Z2D, FFTBatch);
    hipMemset(d_in, 0, FFTBatch * (N + 1) * sizeof(hipfftDoubleComplex));

    int *temp2 = temp_accum_a_b;
    int *temp3 = accum_a_b;

//    int *exp = new int[nBits * N];
//    for (int i = 0; i < nBits * N; ++i) {
//        exp[i] = i << 5 + 1 << 20;
//    }
//    hipMemcpy(temp3, exp, nBits * N * sizeof(int), hipMemcpyHostToDevice);
//    hipMemcpy(temp2, exp, nBits * N * sizeof(int), hipMemcpyHostToDevice);
//    hipMemcpy(temp3 + nBits * N, exp, nBits * N * sizeof(int), hipMemcpyHostToDevice);
//    hipMemcpy(temp2 + nBits * N, exp, nBits * N * sizeof(int), hipMemcpyHostToDevice);

        //create streams
    hipDeviceProp_t cProfile;
    hipGetDeviceProperties(&cProfile, 0);
    int nSM = cProfile.multiProcessorCount;
    cout << "#SM: " << nSM << endl; //20
    hipStream_t streams[nSM];

#pragma unroll
    for (int i = 0; i < 20; ++i) {//nSM
        hipStreamCreateWithFlags(&streams[i], hipStreamNonBlocking);
    }

    ofstream myfile;
    myfile.open ("fullGPU.txt", ios::out);
    for (int j = 0; j < 1; ++j) {

        gridSize = nBits * (k + 1);//as accum is of (k + 1) * 1024;
//        prepareForiFFT_n_Bit<<<gridSize, BLOCKSIZE>>>(temp2, decaCoalesce, d_rev_in, nBits, gridSize, bara, j, temp3);

//
        for (int bIndex = 0; bIndex < nBits; ++bIndex) {
            int temp2Start = bIndex * N * (k + 1);
            int decaCoalesceStart = bIndex * 1024 * kpl;
            int d_rev_inStart = bIndex * 2048 * kpl;
            int baraStart = bIndex * N;
            int nUnitBit = 1;
            gridSize = nUnitBit * (k + 1);//1 * 2
            prepareForiFFT_n_Bit<<<gridSize, BLOCKSIZE>>>(temp2 + temp2Start, decaCoalesce + decaCoalesceStart,
                    d_rev_in + d_rev_inStart, nUnitBit, gridSize, bara + baraStart, j, temp3 + temp2Start);
        }

        hipDeviceSynchronize();

        hipfftExecD2Z(rev_p, d_rev_in, d_rev_out);
        hipDeviceSynchronize();

        int length = nBits * 1 * Ns2;//kpl
        gridSize = (int) ceil((float) (length) / BLOCKSIZE);
        int bkKeyIndex = j * (k + 1) * kpl * Ns2;
        prepareForFFT_n_Bit<<<gridSize, BLOCKSIZE>>>(cuDecaFFTCoalesce, tmpa_gpuCoal, d_in, d_rev_out, cudaBkFFTCoalesceExt, j, nBits, gridSize);

        hipfftExecZ2D(p, d_in, d_out);
        hipDeviceSynchronize();

        length = nBits * N * 2;
        gridSize = (int) ceil((float) (length) / BLOCKSIZE); //2*nBits
        finishUpFFT_n_Bit<<<gridSize, BLOCKSIZE>>>(temp2, d_out, temp3, nBits);

        hipDeviceSynchronize();

//        myfile << "j: " << j << " input: ";
//        length = nBits * N * (k + 1);//nBits * (N + 1) * (k + 1);//iFFTBatch * Ns2;
//        int *temp = new int[length];
//        hipMemcpy(temp, temp3, length * sizeof(int), hipMemcpyDeviceToHost);
//        for (int i = 0; i < nBits * (k + 1); ++i) {
//            int sI = i * N;//(N + s1);
//            for (int j = 0; j < 10; ++j) {
//                myfile << temp[sI + j] << " ";
//            }
////        cout << endl;
//        }
//        myfile << endl;

//        length = nBits * _2N * kpl;
////        myfile << "j: " << j << " output: ";
//        hipfftDoubleReal *temp = new hipfftDoubleReal[length];
//        hipMemcpy(temp, d_rev_in, length * sizeof(hipfftDoubleReal), hipMemcpyDeviceToHost);
//        for (int i = 0; i < nBits * kpl; ++i) {
//            int sI = i * (_2N);
//            for (int j = 0; j < 10; ++j) {
////                myfile << temp[sI + j] << " ";
//                cout << temp[sI + j] << " ";
//            }
//            cout << endl;
//        }
////        myfile << endl;
//        cout << endl;

        swap(temp2, temp3);


//        length = FFTBatch * (N + 1);//nBits * kpl * (N + 1);
//        hipfftDoubleComplex *tempxx = new hipfftDoubleComplex[length];
//        hipMemcpy(tempxx, d_in, length * sizeof(hipfftDoubleComplex), hipMemcpyDeviceToHost);
//        for (int i = 0; i < FFTBatch; ++i) {
//            int sI = i * (N + 1);//(N + 1);//(N + s1);
//            for (int j = 0; j < (N + 1); ++j) {
//                cout << "(" << tempxx[sI + j].x << "," <<  tempxx[sI + j].y << ") ";
//            }
//            cout << endl;
//        }

//make main && ./main 10 2 > test1.txt  && ./main 10 2 >> test1.txt && ./main 10 2 >> test1.txt && ./main 10 2 >> test1.txt  && ./main 10 2 >> test1.txt && ./main 10 2 >> test1.txt && ./main 10 2 >> test1.txt && ./main 10 2 >> test1.txt && ./main 10 2 >> test1.txt && ./main 10 2 >> test1.txt && ./main 10 2 >> test1.txt && ./main 10 2 >> test1.txt && ./main 10 2 >> test1.txt && ./main 10 2 >> test1.txt && ./main 10 2 >> test1.txt && ./main 10 2 >> test1.txt && ./main 10 2 >> test1.txt && ./main 10 2 >> test1.txt && ./main 10 2 >> test1.txt && ./main 10 2 >> test1.txt && ./main 10 2 >> test1.txt && ./main 10 2 >> test1.txt && ./main 10 2 >> test1.txt && ./main 10 2 >> test1.txt && ./main 10 2 >> test1.txt && ./main 10 2 >> test1.txt && ./main 10 2 >> test1.txt && ./main 10 2 >> test1.txt && ./main 10 2 >> test1.txt

    }
//    myfile << endl;
    myfile.close();




    length = nBits * N * (k + 1);//nBits * (N + 1) * (k + 1);//iFFTBatch * Ns2;
    int *temp = new int[length];
    hipMemcpy(temp, temp3, length * sizeof(int), hipMemcpyDeviceToHost);
    for (int i = 0; i < nBits * 2; ++i) {
        int sI = i * N;//(N + 1);
        for (int j = 0; j < 10; ++j) {
            cout << temp[sI + j] << " ";
//                cout << "(" << temp[sI + j].x << "," <<  temp[sI + j].y << ") ";
        }
        cout << endl;
    }





#pragma unroll
    for (int i = 0; i < 20; ++i) { //nSM
        hipStreamDestroy(streams[i]);

    }



    delete [] temp_res_b;
    hipFree(temp_res_a);
    hipFree(accum_a_b);
    hipFree(temp_accum_a_b);
    hipFree(bara);
    hipFree(barb);
    hipFree(testvectbis);

}